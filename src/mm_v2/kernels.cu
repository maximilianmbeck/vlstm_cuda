#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include "../util/cuda_errorcheck.h"
#include "../util/inline_ops.cuh"
#include "../util/inline_print.cuh"
#include "../util/support.h"
#include "kernel_dispatchers.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <cuda/pipeline>
#include <hip/hip_bf16.h>
#include <mma.h>
#include <stdio.h>

// helper functions and utilities to work with CUDA
#include "hip/hip_runtime_api.h"
// #include <helper_functions.h>

/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// CUDA sample demonstrating a __hip_bfloat16 (E8M7) GEMM computation using the
// Warp Matrix Multiply and Accumulate API introduced in CUDA 11.0.

// In this program, the compute_gemm kernel computes the result of a matrix
// multiplication and addition: D = alpha * A * B + beta * C. The dimensions of
// both C and D matrices are M_GLOBAL x N_GLOBAL. The A matrix is M_GLOBAL x
// K_GLOBAL (row-major), the B matrix is K_GLOBAL x N_GLOBAL (column-major). In
// that kernel, each CTA computes one 128 x 128 tile of the resulting matrix per
// iteration. When the tile is computed, the CTA stores it to the global memory
// and begins a new iteration, selecting a new 128 x 128 tile to compute.
// Each CTA consists of eight warps. For the 128 x 128 tile, each warp computes
// eight 16 x 16 subtiles, organized in a 2 x 4 two-dimensional array. Warps
// compute the 16 x 16 subtiles using nvcuda::wmma::mma_sync operations by
// moving through the K_GLOBAL dimension of the A and B matrices and
// accumulating the intermediate result in the local thread state.

// There are a number of simple optimizations used in the algorithm:
// - The CTA copies the 128 x 128 tile of the C matrix from the global memory to
//   shared memory. After that is done, each warp loads the C matrix fragments
//   from shared memory, thus avoiding a random global memory access.
// - On each internal iteration, the CTA copies a portion of the A and B
// matrices from
//   global memory to shared memory. After that, all warps in the CTA reuse the
//   A and B data from shared memory, thus reducing the number of data copies
//   from global memory.
// - The portions of the A and B matrices are stored in shared memory with an
// additional
//   padding (skew) to reduce the number of shared memory access bank conflicts.
//   (See a detailed explanation near the SKEW_BF16 macro definition.)
// - When the CTA finishes computing the tiles of the resulting matrix, each
// warp stores
//   its subtiles to shared memory. The CTA then copies the shared memory
//   contents to global memory, again avoiding redundant random global memory
//   accesses.
// - Note that the CTA tile size is chosen to maximize the GPU register
// utilization,
//   but carefully enough to avoid local memory use.

// Externally configurable parameters.

#ifndef CPU_DEBUG
// Set this to 1 to verify the correctness of the GPU-computed matrix.
#define CPU_DEBUG 0
#endif

#ifndef SHARED_MEMORY_LIMIT_64K
// Set this to 0 to use more than 64 Kb of shared memory to cache data, to
// improve the performance of the computations on GPU.
// Note that you need a GPU that can have more than 64 Kb of shared memory
// per multiprocessor.
#define SHARED_MEMORY_LIMIT_64K 0
#endif

// GPU configuration.

#define WARP_SIZE 32

// MMA matrix tile dimensions.

#define M 16
#define N 16
#define K 16

// GEMM configuration.

#define M_TILES 512
#define N_TILES 512
#define K_TILES 512

#define M_GLOBAL (M * M_TILES)
#define N_GLOBAL (N * N_TILES)
#define K_GLOBAL (K * K_TILES)

#define C_LAYOUT wmma::mem_row_major

// Implementation constants.

#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#if SHARED_MEMORY_LIMIT_64K
// With only 64 Kb shared memory available, we can fit two 8-tile chunks of
// the A and B matrix data, that is (M = 16) * (K = 16) * 8 * (CHUNK_K = 8)
// * sizeof(__hip_bfloat16) = 32 Kb each.
// (i.e. two 8x8 arrays of tiles of 16x16 __hip_bfloat16-typed elements per CTA).
// But we cannot account the 8 Kb total skew overhead, without which the
// performance would be severely impacted. So we choose to reduce the chunk size
// in half, i.e. the amount of A and B matrix data we cache in shared memory.
// Accordingly, this doubles the number of outer iterations across the global K
// dimension, which only slightly impacts the performance.
#define CHUNK_K 4
#else
#define CHUNK_K 8
#endif

#define CHUNK_LINE_BYTES (CHUNK_K * K * sizeof(__hip_bfloat16))
#define WARP_COPY_BYTES (WARP_SIZE * sizeof(int4))
#define CHUNK_COPY_LINES_PER_WARP (WARP_COPY_BYTES / CHUNK_LINE_BYTES)
#define CHUNK_COPY_LINE_LANES (WARP_SIZE / CHUNK_COPY_LINES_PER_WARP)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

#define GLOBAL_MEM_STRIDE N_GLOBAL

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)

// The macro below is used to shift rows of the A matrix and columns of the B
// matrix in shared memory to minimize possible bank conflicts. Before
// performing the nvcuda::wmma::mma_sync operation, the warp must load the
// matrix data using the nvcuda::wmma::load_matrix_sync operation. Although the
// memory access pattern is not specified for that function, each lane in the
// warp can read one or multiple matrix elements from different matrix rows or
// columns. For shared memory, such access can result in bank conflicts if
// different rows / columns of the matrix map to the same bank. By shifting each
// row and column by a few bytes, we make sure that they map to different banks,
// thus reducing the number of possible bank conflicts. The number of 16
// two-byte "__hip_bfloat16" elements is chosen as the minimum possible shift
// because we must keep each row and column 256-bit aligned, as required by
// nvcuda::wmma::load_matrix_sync.
#define SKEW_BF16 16

//! Only for readability purposes. Uncomment
#define __CUDA_ARCH__ 800

#define checkKernelErrors(expr)                                                \
  do {                                                                         \
    expr;                                                                      \
                                                                               \
    hipError_t __err = hipGetLastError();                                    \
    if (__err != hipSuccess) {                                                \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr,                    \
             hipGetErrorString(__err));                                       \
      abort();                                                                 \
    }                                                                          \
  } while (0)

// enum kernels {
//   bf16mma_shmem_gemm_async_copy =
//       0, // __hip_bfloat16 MMA shmem using kernel with async_copy
//   bf16mma_shmem_gemm = 1, // __hip_bfloat16 MMA shmem using kernel normal copy
//                           // (without async_copy).
//   simple_bf16mma_gemm = 2 // __hip_bfloat16 MMA non-shmem using simple kernel.
// };

// const char *kernelNames[] = {"compute_bf16gemm_async_copy",
// "compute_bf16gemm",
//                              "simple_wmma_bf16gemm"};

using namespace nvcuda;

namespace vlstm {

// namespace kernels {

// /*MatrixMul kernel does matrix multiplication from the NVIDA cuda_samples
//  * repo.*/
// template <typename scalar_t, int BLOCKSIZE>
// __global__ void mmkernel(scalar_t *matC, scalar_t *matA, scalar_t *matB, int
// m,
//                          int n, int k);

// } // namespace kernels

__global__ void mmkernel(const __hip_bfloat16 *A, const __hip_bfloat16 *B,
                         const float *C, float *D, float alpha, float beta) {
#if __CUDA_ARCH__ >= 800
  extern __shared__ __hip_bfloat16 shmem[][CHUNK_K * K + SKEW_BF16];

  // Warp and lane identification.
  const unsigned int warpId = threadIdx.x / WARP_SIZE;
  const unsigned int laneId = threadIdx.x % WARP_SIZE;

  // Offset in shared memory from which the B matrix is stored.
  const size_t shmem_idx_b_off = BLOCK_COL_TILES * M;

  // This pointer is used to access the C and D matrix tiles this warp computes.
  float *shmem_warp_tile_ptr =
      (float *)&shmem[0][0] +
      (warpId / BLOCK_ROW_WARPS) * SHMEM_STRIDE * N * BLOCK_ROW_WARPS +
      (warpId % BLOCK_ROW_WARPS) * SHMEM_OFFSET;

  // This pointer is used to stream the C and D matrices block-wide tile to and
  // from shared memory.
  float *shmem_warp_stream_ptr =
      (float *)&shmem[0][0] + warpId * SHMEM_STRIDE * N;

  // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
  // each tile computation. Technically this is not generally correct (may
  // result in a loss of precision). Zero still needs to be specially handled
  // though.
  beta /= alpha;

  // Each CTA slides along the 128 x 128 tiles from the top left corner of the
  // matrix to the right and down, and selects the next tile to compute. Once
  // there's no such tile, all warps in this CTA exit.
  for (unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
    const unsigned int block_tile_i =
        ((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
    const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;

    // Stop when there are no more D matrix tiles to compute in this CTA.
    if (block_tile_i >= M_TILES) {
      break;
    }

    // This warp's pointer to the C matrix data to copy memory from to shared
    // memory.
    const size_t gmem_idx =
        (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;
    const float *src_gmem_warp_stream_ptr = &C[gmem_idx];

    // Stream multiple C tiles to shared memory.
#pragma unroll
    for (int i = 0; i < N; i++) {
      *((int4 *)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId) = *(
          (int4 *)(src_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId);
    }

    __syncthreads();

    // These fragments will accumulate the result of A and B matrix fragment
    // multiplications along the K_GLOBAL dimension.
    wmma::fragment<wmma::accumulator, M, N, K, float> c[WARP_COL_TILES]
                                                       [WARP_ROW_TILES];

    // Load the C matrix tiles into fragments from shared memory.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
        const float *tile_ptr =
            shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;

        wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE, C_LAYOUT);
      }
    }

    __syncthreads();

    // Scale the C matrix.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
        for (int t = 0; t < c[i][j].num_elements; t++) {
          c[i][j].x[t] *= beta;
        }
      }
    }

    // Select what warp copies what matrix to shared memory.
    // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
    const __hip_bfloat16 *warp_ptr =
        (warpId < (WARPS_PER_BLOCK / 2))
            ? (&A[block_tile_i * M * K_GLOBAL] +
               M * K_GLOBAL * (warpId % (WARPS_PER_BLOCK / 2)) * 2)
            : (&B[block_tile_j * N * K_GLOBAL] +
               N * K_GLOBAL * (warpId % (WARPS_PER_BLOCK / 2)) * 2);

    // Go through the global K dimension by a fixed step at a time.
#pragma unroll
    for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
      // Copy slices of the A and B matrices to shared memory.
      // The first half of the warps in the CTA copy the A matrix, the rest copy
      // the B matrix.
      size_t shmem_idx =
          warpId < (WARPS_PER_BLOCK / 2)
              ? (M * (warpId % (WARPS_PER_BLOCK / 2)) * 2)
              : (N * (warpId % (WARPS_PER_BLOCK / 2)) * 2 + shmem_idx_b_off);

      // First half of the warp copies the first row / column of the matrix,
      // the second half of the warp copies the next.
      const __hip_bfloat16 *lane_ptr =
          (warp_ptr + tile_k * K + (laneId / CHUNK_COPY_LINE_LANES) * K_GLOBAL);

      // Shift the second half of the warp to the next row / column in the
      // shared memory.
      shmem_idx += laneId / CHUNK_COPY_LINE_LANES;

#pragma unroll
      for (int i = 0; i < ((WARP_SIZE / 2) / CHUNK_COPY_LINES_PER_WARP) * 2;
           i++) {
        // Copy 16 bytes at once in each lane.
        *((int4 *)&shmem[shmem_idx][0] + (laneId % CHUNK_COPY_LINE_LANES)) =
            *((int4 *)lane_ptr + (laneId % CHUNK_COPY_LINE_LANES));

        // Advance the global memory pointer and the shared memory index.
        lane_ptr = lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP;
        shmem_idx += CHUNK_COPY_LINES_PER_WARP;
      }

      __syncthreads();

      // Compute a grid of C matrix tiles in each warp.
#pragma unroll
      for (int k_step = 0; k_step < CHUNK_K; k_step++) {
        wmma::fragment<wmma::matrix_a, M, N, K, __hip_bfloat16, wmma::row_major>
            a[WARP_COL_TILES];
        wmma::fragment<wmma::matrix_b, M, N, K, __hip_bfloat16, wmma::col_major>
            b[WARP_ROW_TILES];

#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
          size_t shmem_idx_a =
              (warpId / BLOCK_ROW_WARPS) * M * BLOCK_ROW_WARPS + (i * M);
          const __hip_bfloat16 *tile_ptr = &shmem[shmem_idx_a][k_step * K];

          wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_BF16);

#pragma unroll
          for (int j = 0; j < WARP_ROW_TILES; j++) {
            if (i == 0) {
              // Load the B matrix fragment once, because it is going to be
              // reused against the other A matrix fragments.
              size_t shmem_idx_b = shmem_idx_b_off +
                                   (WARP_ROW_TILES * N) * (warpId % 2) +
                                   (j * N);
              const __hip_bfloat16 *tile_ptr = &shmem[shmem_idx_b][k_step * K];

              wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_BF16);
            }

            wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
          }
        }
      }

      __syncthreads();
    }

    // Store the D fragments to shared memory.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
        // Uniform, point-wise transformations of ALL fragment elements by ALL
        // threads in the warp are well-defined even though element indices
        // within fragment storage are not defined.
        for (int t = 0; t < c[i][j].num_elements; t++)
          c[i][j].x[t] *= alpha;

        float *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * K + j * N;

        wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
      }
    }

    __syncthreads();

    // Now that shared memory contains all the D tiles, stream them to global
    // memory.
    float *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
    for (int i = 0; i < N; i++) {
      *((float4 *)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
          *((float4 *)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
    }

    __syncthreads();
  }
#endif
}

/*
A: (m x k)
B: (k x n)
C: (m x n)
*/
template <typename scalar_t>
void kernel_dispatchers::mmkernel_dispatch(float *matD, scalar_t *matA,
                                           scalar_t *matB, float *matC, int m,
                                           int n, int k) {
  // printf("m: %d, n: %d, k: %d\n", m, n, k);

  enum {
    // Compute the right amount of shared memory to request.
    // We need shared memory to hold per-CTA C and D matrix tiles, and to cache
    // per-CTA chunks
    // of the A and B matrices. Therefore, the right amount to request is the
    // maximum of those
    // two numbers.
    SHMEM_SZ = MAX(sizeof(__hip_bfloat16) * (BLOCK_COL_TILES * M) *
                       (CHUNK_K * K + SKEW_BF16) * 2,
                   M * (BLOCK_ROW_WARPS * WARP_ROW_TILES) * N *
                       (BLOCK_COL_WARPS * WARP_COL_TILES) * sizeof(float))
  };
  uint shmem_sz_1 = sizeof(__hip_bfloat16) * (BLOCK_COL_TILES * M) *
                    (CHUNK_K * K + SKEW_BF16) * 2;
  uint shmem_sz_2 = M * (BLOCK_ROW_WARPS * WARP_ROW_TILES) * N *
                    (BLOCK_COL_WARPS * WARP_COL_TILES) * sizeof(float);
  printf("Required shared memory size: %d bytes = MAX(%d, %d)\n", SHMEM_SZ,
         shmem_sz_1, shmem_sz_2);
  printf("Required shared memory size: %lu Kb\n", SHMEM_SZ / 1024UL);

  printf("#bytes: sizeof(int4)= %lu, sizeof(float4)= %lu, "
         "sizeof(__hip_bfloat16)= %lu\n",
         sizeof(int4), sizeof(float4), sizeof(__hip_bfloat16));
  printf("CHUNK_K=%d, CHUNK_LINE_BYTES=%lu, WARP_COPY_BYTES=%lu, "
         "CHUNK_COPY_LINES_PER_WARP=%lu, CHUNK_COPY_LINE_LANES=%lu\n",
         CHUNK_K, CHUNK_LINE_BYTES, WARP_COPY_BYTES, CHUNK_COPY_LINES_PER_WARP,
         CHUNK_COPY_LINE_LANES);
  printf("BLOCK_ROW_TILES=%d, BLOCK_COL_TILES=%d, BLOCK_ROW_WARPS=%d, "
         "BLOCK_COL_WARPS=%d\n",
         BLOCK_ROW_TILES, BLOCK_COL_TILES, BLOCK_ROW_WARPS, BLOCK_COL_WARPS);
  printf("WARP_ROW_TILES=%d, WARP_COL_TILES=%d, WARP_SIZE=%d, "
         "WARPS_PER_BLOCK=%d\n",
         WARP_ROW_TILES, WARP_COL_TILES, WARP_SIZE, WARPS_PER_BLOCK);
  printf("SHMEM_STRIDE=%d, SHMEM_OFFSET=%d\n", SHMEM_STRIDE, SHMEM_OFFSET);
  const float alpha = 1.1f;
  const float beta = 1.2f;

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, 0));

  checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(
      mmkernel), hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ));

  checkKernelErrors(
      (mmkernel<<<deviceProp.multiProcessorCount * 2, THREADS_PER_BLOCK,
                  SHMEM_SZ>>>(matA, matB, matC, matD, alpha, beta)));

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());
}

template void kernel_dispatchers::mmkernel_dispatch<__hip_bfloat16>(
    float *matD, __hip_bfloat16 *matA, __hip_bfloat16 *matB, float *matC, int m,
    int n, int k);
// template void kernel_dispatchers::mmkernel_dispatch<__half>(__half *matC,
//                                                             __half *matA,
//                                                             __half *matB, int
//                                                             m, int n, int k);

} // namespace vlstm