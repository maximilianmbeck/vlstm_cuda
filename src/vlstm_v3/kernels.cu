#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>

#include "../util/cuda_errorcheck.h"
#include "../util/inline_ops.cuh"
#include "../util/inline_print.cuh"
#include "../util/support.h"
#include "kernel_dispatchers.h"

#define CEIL_DIV(a, b) (((a) + (b)-1) / (b))
#define MAX(a, b) (((a) > (b)) ? (a) : (b))
#define MIN(a, b) (((a) < (b)) ? (a) : (b))

namespace vlstm {

namespace kernels {

/* MatrixMul kernel does matrix multiplication from the NVIDA cuda_samples
 repo.*/
template <typename scalar_t, int BLOCKSIZE>
__global__ void mmkernelv1(scalar_t *matC, scalar_t *matA, scalar_t *matB,
                           int m, int n, int k);

template <typename scalar_t, int TblockDim>
__global__ void qkvkernel(scalar_t *matC, scalar_t *matQ, scalar_t *matK,
                          scalar_t *matV, int batchSize, int numHeads,
                          int seqLen, int dimHeads);

} // namespace kernels

////////////////////////////////////////////////////////////////////////////////////////
/* MATRIXMUL KERNEL V1*/
/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * k is A's width and n is B's width
 * works only if m, n, k are divisible by BLOCK_SIZE
 */
template <typename scalar_t, int BLOCK_SIZE>
__global__ void kernels::mmkernelv1(scalar_t *C, scalar_t *A, scalar_t *B,
                                    int m, int n, int k) {
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  // int tIdx = threadIdx.x + blockDim.x * threadIdx.y;

  int cx = bx * blockDim.x + tx;
  int cy = by * blockDim.y + ty;
  if ((cx == 0) && (cy == 0)) {
    printf("In Kernel: m: %d, n: %d, k: %d\n", m, n, k);
    printf("In Kernel: gdim.x: %d, gdim.y: %d, bdim.x: %d, bdim.y: %d\n",
           gridDim.x, gridDim.y, blockDim.x, blockDim.y);
  }
  if (false) {
    // printf("bdx: %d, bdy: %d, b_cx: %d, b_cy: %d, tIdx: %d\n", blockDim.x,
    //        blockDim.y, bx * blockDim.x + tx, by * blockDim.y + ty, tIdx);
    printf("cx: %d, cy: %d\n", cx, cy);
  }
  // @max Output index of the matrix C
  int block_cIdx = n * BLOCK_SIZE * by + BLOCK_SIZE * bx; // (blocklevel)
  int thread_cIdx = block_cIdx + n * ty + tx;             // (threadlevel)

  // Index of the first sub-matrix of A processed by the block
  int aBegin = k * BLOCK_SIZE * by; // (blocklevel)

  // Index of the last sub-matrix of A processed by the block
  int aEnd = aBegin + k - 1; // (blocklevel)

  // Step size used to iterate through the sub-matrices of A
  int aStep = BLOCK_SIZE; // (blocklevel)

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx; // (blocklevel)

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE * n; // (blocklevel)

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  // @max: Csub is also used to accumulate the result for one entry in the
  // output matrix C
  scalar_t Csub = dscalar_zero<scalar_t>();

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  // @max: outer loop, progresses always in BLOCK_SIZE steps and accumulates
  // the final values in Csub
  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ scalar_t As[BLOCK_SIZE][BLOCK_SIZE];
    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ scalar_t Bs[BLOCK_SIZE][BLOCK_SIZE];
    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix

    // @max: There are three bounds to consider:
    // TODO rewrite kernel with true indices not with memory offsets
    // - m & n in the B matrix: the tread block bounds
    // - k in the A and B matrix: the bounds of the loop over the k dimension
    // TODO @max: WRONG! the bounds of the loop over the k dimension are not
    // correct! rewrite!
    As[ty][tx] = A[a + k * ty + tx]; // (threadlevel)
    Bs[ty][tx] = B[b + n * ty + tx]; // (threadlevel)
    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll
    // @max: inner looop
    for (int i = 0; i < BLOCK_SIZE; ++i) {
      if ((cx == 0) && (cy == 9)) {
        print_val("(ty,i) - As", ty, i, As[ty][i]);
        print_val("(ty,i) - Bs", ty, i, Bs[ty][i]);
      }
      Csub = add_g(Csub, mul_g(As[ty][i],
                               Bs[i][tx])); // (threadlevel): each thread
                                            // operates on BLOCK_SIZE elements
      if ((cx == 0) && (cy == 9)) {
        print_val("(cx,cy)-InLoop:Csub", cx, cy, Csub);
      }
    }
    if ((cx == 0) && (cy == 9)) {
      print_val("(cx,cy)-AfterLoop:Csub", cx, cy, Csub);
    }
    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }
  // Write the block sub-matrix to device memory;
  // each thread writes one element
  C[thread_cIdx] = Csub; // (threadlevel)
}

/*
A: (m x k)
B: (k x n)
C: (m x n)
*/
template <typename scalar_t>
void kernel_dispatchers::mmkernelv1_dispatch(scalar_t *matC, scalar_t *matA,
                                             scalar_t *matB, int m, int n,
                                             int k) {
  printf("m: %d, n: %d, k: %d\n", m, n, k);
  const int BLOCK_SIZE = 4;

  // determine the number of blocks and threads
  const dim3 blockDims(BLOCK_SIZE, BLOCK_SIZE);
  const dim3 gridDims((n + blockDims.x - 1) / blockDims.x,
                      (m + blockDims.y - 1) / blockDims.y);
  printf("blocksxy: %d-%d, threads: %d-%d\n", gridDims.x, gridDims.y,
         blockDims.x, blockDims.y);

  // if (m % BLOCK_SIZE != 0 || n % BLOCK_SIZE != 0 || k % BLOCK_SIZE != 0) {
  //     printf("m, n, k must be divisible by BLOCK_SIZE\n");
  //     return;
  // }
  kernels::mmkernelv1<scalar_t, BLOCK_SIZE>
      <<<gridDims, blockDims>>>(matC, matA, matB, m, n, k);

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());
}

template void kernel_dispatchers::mmkernelv1_dispatch<__hip_bfloat16>(
    __hip_bfloat16 *matC, __hip_bfloat16 *matA, __hip_bfloat16 *matB, int m, int n,
    int k);
template void kernel_dispatchers::mmkernelv1_dispatch<__half>(
    __half *matC, __half *matA, __half *matB, int m, int n, int k);

////////////////////////////////////////////////////////////////////////////////////////

#define TBLOCK_DIM 4 // TblockDim: corresponds to BLOCK_DIM in matmul

#define DEBUG 1

/* QKV Kernel v1 */

template <typename scalar_t, int TblockDim>
__global__ void kernels::qkvkernel(scalar_t *matC, scalar_t *matQ,
                                   scalar_t *matK, scalar_t *matV,
                                   int batchSize, int numHeads, int seqLen,
                                   int dimHeads) {
  // int tIdx = threadIdx.x + blockDim.x * threadIdx.y;
#ifdef DEBUG
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    printf("In Kernel: gdim.x: %d, gdim.y: %d, bdim.x: %d, bdim.y: %d\n",
           gridDim.x, gridDim.y, blockDim.x, blockDim.y);
  }
#endif

  // Most outer loop: Loop over batchSize * numHeads (can be parallelized later
  // along gridDim.z)
  const uint batchHeadStep = seqLen * dimHeads;
  const uint batchHeadEnd = batchSize * numHeads * batchHeadStep;
  for (uint batchHeadIdx = 0; batchHeadIdx < batchHeadEnd;
       batchHeadIdx += batchHeadStep) {

    // access to Q (copy to C, no transpose)
    const uint qBlockIdx = batchHeadIdx + dimHeads * TblockDim * blockIdx.y +
                           TblockDim * blockIdx.x;
    const uint qThreadIdx = qBlockIdx + TblockDim * threadIdx.y + threadIdx.x;

    // access to K (copy to C, with transpose)
    const uint kBlockIdx =
        batchHeadIdx + seqLen * TblockDim * blockIdx.y + TblockDim * blockIdx.x;
    const uint kThreadIdx = qBlockIdx + TblockDim * threadIdx.x + threadIdx.y;

    matC[qThreadIdx] = matQ[kThreadIdx];

    __syncthreads();

    // const uint cx = bx * blockDim.x + tx;
    // const uint cy = by * blockDim.y + ty;

    // loop over batchSize * numHeads
    // if ((cx == 0) && (cy == 0)) {
    //   printf("In Kernel: m: %d, n: %d, k: %d\n", m, n, k);
    //   printf("In Kernel: gdim.x: %d, gdim.y: %d, bdim.x: %d, bdim.y: %d\n",
    //          gridDim.x, gridDim.y, blockDim.x, blockDim.y);
    // }
    // if (false) {
    //   // printf("bdx: %d, bdy: %d, b_cx: %d, b_cy: %d, tIdx: %d\n",
    //   blockDim.x,
    //   //        blockDim.y, bx * blockDim.x + tx, by * blockDim.y + ty,
    //   tIdx); printf("cx: %d, cy: %d\n", cx, cy);
    // }
    // int block_cIdx = n * TblockDim * by + TblockDim * bx; // (blocklevel)
    // int thread_cIdx = block_cIdx + n * ty + tx;           // (threadlevel)

    //! Comment out the whole computation, at just load Q and store it in C
    //   int aBegin = k * TblockDim * by; // (blocklevel)
    //   int aEnd = aBegin + k; // (blocklevel)
    //   int aStep = TblockDim; // (blocklevel)
    //   int bBegin = TblockDim * bx; // (blocklevel)
    //   int bStep = TblockDim * n; // (blocklevel)

    //   // Csub is used to store the element of the block sub-matrix
    //   // that is computed by the thread
    //   // @max: Csub is also used to accumulate the result for one entry in
    //   the
    //   // output matrix C
    //   scalar_t Csub = dscalar_zero<scalar_t>();

    //   // Loop over all the sub-matrices of A and B
    //   // required to compute the block sub-matrix
    //   // @max: outer loop, progresses always in TblockDim steps and
    //   accumulates
    //   // the final values in Csub
    //   for (int a = aBegin, b = bBegin; a < aEnd; a += aStep, b += bStep) {
    //     __shared__ scalar_t As[TblockDim][TblockDim];
    //     __shared__ scalar_t Bs[TblockDim][TblockDim];

    //     As[ty][tx] = A[a + k * ty + tx]; // (threadlevel)
    //     Bs[ty][tx] = B[b + n * ty + tx]; // (threadlevel)
    //     // Synchronize to make sure the matrices are loaded
    //     __syncthreads();

    // #pragma unroll
    //     for (int i = 0; i < TblockDim; ++i) {
    //       if ((cx == 0) && (cy == 9)) {
    //         print_val("(ty,i) - As", ty, i, As[ty][i]);
    //         print_val("(ty,i) - Bs", ty, i, Bs[ty][i]);
    //       }
    //       Csub = add_g(Csub, mul_g(As[ty][i],
    //                                Bs[i][tx])); // (threadlevel): each thread
    //                                             // operates on TblockDim
    //                                             elements
    //       if ((cx == 0) && (cy == 9)) {
    //         print_val("(cx,cy)-InLoop:Csub", cx, cy, Csub);
    //       }
    //     }
    //     if ((cx == 0) && (cy == 9)) {
    //       print_val("(cx,cy)-AfterLoop:Csub", cx, cy, Csub);
    //     }
    //     __syncthreads();
    //   }
    // matC[thread_cIdx] = Csub; // (threadlevel)
  }
}

template <typename scalar_t>
void kernel_dispatchers::qkvkernel_dispatch(scalar_t *matC, scalar_t *matQ,
                                            scalar_t *matK, scalar_t *matV,
                                            int batchSize, int numHeads,
                                            int seqLen, int dimHeads) {
  printf("B: %d, NH: %d, S: %d, DH: %d\n", batchSize, numHeads, seqLen,
         dimHeads);
  const int TblockDim = TBLOCK_DIM; // Block

  // determine the number of blocks and threads
  const dim3 blockDims(TblockDim, TblockDim);

  const dim3 gridDims(CEIL_DIV(dimHeads, blockDims.x),
                      CEIL_DIV(seqLen, blockDims.y));
  printf("blocksxy: %d-%d, threads: %d-%d\n", gridDims.x, gridDims.y,
         blockDims.x, blockDims.y);

  kernels::qkvkernel<scalar_t, TblockDim><<<gridDims, blockDims>>>(
      matC, matQ, matK, matV, batchSize, numHeads, seqLen, dimHeads);

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());
}

// this is needed to make sure that the compiler instantiates the template
template void kernel_dispatchers::qkvkernel_dispatch<__hip_bfloat16>(
    __hip_bfloat16 *matC, __hip_bfloat16 *matQ, __hip_bfloat16 *matK,
    __hip_bfloat16 *matV, int batchSize, int numHeads, int seqLen, int dimHeads);
template void kernel_dispatchers::qkvkernel_dispatch<__half>(
    __half *matC, __half *matQ, __half *matK, __half *matV, int batchSize,
    int numHeads, int seqLen, int dimHeads);

} // namespace vlstm