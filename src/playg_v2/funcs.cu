#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>

#include "../util/support.h"
#include "../util/inline_ops.cuh"
#include "../util/cuda_errorcheck.h"
#include "kernel_dispatchers.h"

namespace vlstm {

namespace kernels {

/*A kernel that copies from A to B*/
template <typename scalar_t>
__global__ void copykernel(scalar_t *mat_A, scalar_t *mat_B, int r,
                           int c);

/*MatrixMul kernel does matrix multiplication from the NVIDA cuda_samples
 * repo.*/
template<int> __global__ void mmkernelv1(__hip_bfloat16 *matC, __hip_bfloat16 *matA,
                           __hip_bfloat16 *matB, int wA, int wB);

} // namespace kernels

/* COPYKERNEL */
template <typename scalar_t>
__global__ void kernels::copykernel<scalar_t>(scalar_t *mat_A, scalar_t *mat_B,
                                    int rdim, int cdim) {
  int cidx = blockIdx.x * blockDim.x + threadIdx.x;
  int ridx = blockIdx.y * blockDim.y + threadIdx.y;

  if (cidx < cdim && ridx < rdim) {
    int idx = ridx + cidx * rdim;
    float val = to_float<scalar_t>(mat_A[idx]);
    printf("cidx: %d, ridx: %d, val: %f\n", cidx, ridx, val);
    mat_B[idx] = mat_A[idx];
  }
}

template __global__ void kernels::copykernel<__hip_bfloat16>(__hip_bfloat16 *mat_A, __hip_bfloat16 *mat_B, int rdim, int cdim);
template __global__ void kernels::copykernel<__half>(__half *mat_A, __half *mat_B, int rdim, int cdim);

template <typename scalar_t> // todo remove this scalar_t after func name
void kernels::copykernel_dispatch<scalar_t>(scalar_t *mat_A, scalar_t *mat_B,
                                  int rows, int cols) {
  printf("rows: %d, cols: %d\n", rows, cols);
  // determine the number of blocks and threads
  const dim3 block_threads(32, 32);
  const dim3 grid_blocks((rows + block_threads.y - 1) / block_threads.y,
                         (cols + block_threads.x - 1) / block_threads.x);
  printf("blocksxy: %d-%d, threads: %d-%d\n", grid_blocks.x, grid_blocks.y,
         block_threads.x, block_threads.y);
  kernels::copykernel<scalar_t><<<grid_blocks, block_threads>>>(mat_A, mat_B, rows, cols);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
}

template void kernels::copykernel_dispatch<__hip_bfloat16>(__hip_bfloat16 *mat_A, __hip_bfloat16 *mat_B, int rows, int cols);
template void kernels::copykernel_dispatch<__half>(__half *mat_A, __half *mat_B, int rows, int cols);

/* MATRIXMUL KERNEL */
/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <typename scalar_t, int BLOCK_SIZE>
__global__ void kernels::mmkernelv1(scalar_t *C, scalar_t *A,
                                    scalar_t *B, int wA, int wB) {
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  scalar_t Csub = dscalar_zero<scalar_t>();

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ scalar_t As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ scalar_t Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub = add_g(Csub, mul_g(As[ty][k], Bs[k][tx]));
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;
}

/*
A: (m x k)
B: (k x n)
C: (m x n)
*/
template <typename scalar_t>
void kernels::mmkernelv1_dispatch(scalar_t *matC, scalar_t *matA,
                                  scalar_t *matB, int m, int n, int k) {
  const int BLOCK_SIZE = 32;

  printf("m: %d, n: %d, k: %d\n", m, n, k);
  if (m % BLOCK_SIZE != 0 || n % BLOCK_SIZE != 0 || k % BLOCK_SIZE != 0) {
      printf("m, n, k must be divisible by BLOCK_SIZE\n");
      return;
    }

  // determine the number of blocks and threads
  const dim3 block_threads(BLOCK_SIZE, BLOCK_SIZE);
  const dim3 grid_blocks((m + block_threads.y - 1) / block_threads.y,
                         (n + block_threads.x - 1) / block_threads.x);
  printf("blocksxy: %d-%d, threads: %d-%d\n", grid_blocks.x, grid_blocks.y,
         block_threads.x, block_threads.y);
  kernels::mmkernelv1<scalar_t, BLOCK_SIZE>
      <<<grid_blocks, block_threads>>>(matC, matA, matB, k, n);

  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

}

template void kernels::mmkernelv1_dispatch<__hip_bfloat16>(__hip_bfloat16 *matC, __hip_bfloat16 *matA, __hip_bfloat16 *matB, int m, int n, int k);
template void kernels::mmkernelv1_dispatch<__half>(__half *matC, __half *matA, __half *matB, int m, int n, int k);

} // namespace vlstm