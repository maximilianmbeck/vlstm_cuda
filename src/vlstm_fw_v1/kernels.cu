#include "hip/hip_runtime.h"
// Copyright JKU Linz 2023
// Author: Maximilian Beck

#include <hip/hip_cooperative_groups.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <hip/hip_math_constants.h>

#include "../util/cuda_errorcheck.h"
#include "../util/inline_ops.cuh"
#include "../util/inline_print.cuh"
#include "../util/support.h"
#include "kernel_dispatchers.h"

#define CEIL_DIV(a, b) (((a) + (b)-1) / (b))
#define MAX(a, b) (((a) > (b)) ? (a) : (b))
#define MIN(a, b) (((a) < (b)) ? (a) : (b))

namespace vlstm {

namespace cg = cooperative_groups;

namespace kernels {

template <typename scalar_t, int TblockDim, int QblockDim, int KVblockDim>
__global__ void vlstm_fw(scalar_t *matH, scalar_t *matC, scalar_t *matQ,
                         scalar_t *matK, scalar_t *matV, scalar_t *iGatePreact,
                         scalar_t *fGatePreact, int batchSize, int numHeads,
                         int seqLen, int dimHeads);

} // namespace kernels

////////////////////////////////////////////////////////////////////////////////////////

#define TBLOCK_DIM 4 // TblockDim: corresponds to BLOCK_DIM in matmul
#define KVTILE_DIM 8 // KVtileDim: TileDim for K&V along seqLen dim
// QTILE_DIM must be divisible by KVTILE_DIM and TBLOCK_DIM
#define QTILE_DIM 8 // QtileDim: TileDim for Q along seqLen dim

// shared memory must be aligned: depends on scalar_t (multiples of 4 should be
// fine for bf16, fp16 and fp32)
#define SHARED_MEM_PADDING 8 // SHARED_MEM_PADDING: padding for shared memory

// SMEMARRAY: access shared memory array (2D)
#define SMEMARRAY(array, stride, row, col)                                     \
  array[(row) * (stride + SHARED_MEM_PADDING) + (col)]
// SMEMVECTOR: access shared memory vector (1D)
#define SMEMVECTOR(array, idx) array[(idx) * (1 + SHARED_MEM_PADDING)]

#define DEBUG 1
// #define DEBUG2 1
// #define DEBUG3 1
// #define DEBUG4 1
// #define DEBUG5 1

/**
Conventions:
- chunk: A 1D vector in shared memory
- tile: A 2D matrix in shared memory

*/

/* vLSTM Forward Kernel v0 */

template <typename scalar_t, int TblockDim, int QtileDim, int KVtileDim>
__global__ void kernels::vlstm_fw(scalar_t *matH, scalar_t *matC,
                                  scalar_t *matQ, scalar_t *matK,
                                  scalar_t *matV, scalar_t *iGatePreact,
                                  scalar_t *fGatePreact, int batchSize,
                                  int numHeads, int seqLen, int dimHeads) {
  // int tIdx = threadIdx.x + blockDim.x * threadIdx.y;
#ifdef DEBUG
  if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
      (threadIdx.y == 0)) {
    printf("In Kernel: gdim.x: %d, gdim.y: %d, gdim.z: %d, bdim.x: %d, bdim.y: "
           "%d\n",
           gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y);
  }
#endif

#ifdef DEBUG
  if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
      (threadIdx.y == 0)) {
    printf("In Kernel: QtileDim: %d, KVtileDim: %d, TblockDim:%d\n", QtileDim,
           KVtileDim, TblockDim);
  }
#endif
  cg::grid_group gridGroup = cg::this_grid();

  //! Shared Memory aka SRAM
  // the data in this shared memory is shared across all threads in a thread
  // block
  extern __shared__ float sbuf[]; // declare it as float and redefine it later

  //? for inputs
  // Note: keep in mind the memory is defined in a contiguous region.
  // One pointer has the full memory space until the next point is defined.
  // Therefore to read the size of a single shared memory array you need to
  // have a look at the offset for the next array.

  // qtile (QtileDim x dimHeads) in shared memory (padding for alignment)
  scalar_t *qTile = (scalar_t *)sbuf;
  // kTile and vTile (KVtileDim x dimHeads) in shared memory
  scalar_t *kTile =
      (scalar_t *)&qTile[QtileDim * (dimHeads + SHARED_MEM_PADDING)];
  scalar_t *vTile =
      (scalar_t *)&kTile[KVtileDim * (dimHeads + SHARED_MEM_PADDING)];

  //? for intermediate results
  // init cTile (QtileDim x KVTileDim) in shared memory for intermediate
  // result of QK^T
  scalar_t *cTile =
      (scalar_t *)&vTile[KVtileDim * (dimHeads + SHARED_MEM_PADDING)];
  // init result hTile (QTileDim x dimHeads) in shared memory
  scalar_t *hTile =
      (scalar_t *)&cTile[QtileDim * (KVtileDim + SHARED_MEM_PADDING)];
  // init dTile (QTileDim x KVTileDim) in shared memory for forget and input
  // gate matrix
  scalar_t *dTile =
      (scalar_t *)&hTile[QtileDim * (dimHeads + SHARED_MEM_PADDING)];

  //? for input and forget gate
  // init iChunk (KVTileDim x 1) in shared memory for input gate
  scalar_t *iChunk =
      (scalar_t *)&dTile[QtileDim * (dimHeads + SHARED_MEM_PADDING)];
  // init fChunk (QTileDim x 1) in shared memory for forget gate
  scalar_t *fChunk = (scalar_t *)&iChunk[KVtileDim * (1 + SHARED_MEM_PADDING)];
  // init fTileCol (QTileDim x 1) in shared memory for forget gate (first column
  // of the QtileDim x KVtileDim dTile)
  float *fTileCol = (float *)&fChunk[QtileDim * (1 + SHARED_MEM_PADDING)];

  // init mChunk (QTileDim x 1) in shared memory for max state of
  // dTile
  float *mChunk = (float *)&fTileCol[QtileDim * (1 + SHARED_MEM_PADDING)];
  // init mPrevTileCol (QTileDim x 1) in shared memory for previous
  // max state of dTile
  float *mPrevChunk = (float *)&mChunk[QtileDim * (1 + SHARED_MEM_PADDING)];
  // init lChunk (QTileDim x 1) in shared memory for rowsum of cTile * dTile
  float *lChunk = (float *)&mPrevChunk[QtileDim * (1 + SHARED_MEM_PADDING)];
  // init lPrevChunk (QTileDim x 1) in shared memory for previous rowsum of
  // cTile * dTile
  float *lPrevChunk = (float *)&lChunk[QtileDim * (1 + SHARED_MEM_PADDING)];
  // init fTileColLast (1 x 1) in shared memory for forget gate (last row value
  // of fTileCol)
  float *fTileColLast =
      (float *)&lPrevChunk[QtileDim * (1 + SHARED_MEM_PADDING)];

  //! PARALLELIZE ALONG BATCHSIZE * NUMHEADS (gridDim.x)
  const uint batchHeadStepQKV = seqLen * dimHeads;
  const uint batchHeadStepIFgate = seqLen * 1;
  const uint numBatchHeads = batchSize * numHeads;
  // End for looplevel 0:
  const uint batchHeadEnd = CEIL_DIV(numBatchHeads, gridDim.x);
  // looplevel 0: loop over batches and heads
  for (uint batchHeadIdx = 0; batchHeadIdx < batchHeadEnd; ++batchHeadIdx) {

    uint batchHeadGridXGlobalMemIdxQKV =
        (batchHeadStepQKV * gridDim.x) * batchHeadIdx +
        (batchHeadStepQKV)*blockIdx.x;

    uint batchHeadGridXGlobalMemIdxIFgate =
        (batchHeadStepIFgate * gridDim.x) * batchHeadIdx +
        (batchHeadStepIFgate)*blockIdx.x;

#ifdef DEBUG5
    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
      printf("B<%d,%d> batchHeadIdx: %d, batchHeadEnd: %d, "
             "batchHeadGridXGlobalMemIdxQKV: "
             "%d\n",
             blockIdx.x, blockIdx.y, batchHeadIdx, batchHeadEnd,
             batchHeadGridXGlobalMemIdxQKV);
    }
#endif
    SMEMVECTOR(fTileColLast, 0) =
        float2type<float>(0.0f); // could also just write 0.0f

    //! PARALLELIZE ALONG SEQLEN (gridDim.y)
    // Ends for looplevel 1:
    const uint qTileEnd = CEIL_DIV(seqLen, QtileDim * gridDim.y);
    // looplevel 1: loop over Qtile blocks along seqLen dim
    for (uint qTileIdx = 0; qTileIdx < qTileEnd; ++qTileIdx) {

      //* qTile Global Memory Index
      // (grid&block) offset in Q matrix for qTile (global memory)
      const uint qTileGridXYGlobalMemIdx =
          batchHeadGridXGlobalMemIdxQKV +
          (dimHeads * QtileDim * gridDim.y) * qTileIdx;
      const uint qTileBlockGlobalMemIdx =
          qTileGridXYGlobalMemIdx + (dimHeads * QtileDim) * blockIdx.y;

      //* cTile Global Memory Index (virtual, as never materialized fully)
      // (grid&block) offset Y-axis in C = Q*K^T matrix (along sequence
      // dimension) (used for checking causality)
      const uint cTileGridYIdx = QtileDim * gridDim.y * qTileIdx;
      const uint cTileBlockYIdx = cTileGridYIdx + QtileDim * blockIdx.y;

#ifdef DEBUG5
      if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        printf("B<%d,%d> qTileIdx: %d, qTileEnd: %d, "
               "qTileBlockGlobalMemIdx: "
               "%d, \n",
               blockIdx.x, blockIdx.y, qTileIdx, qTileEnd,
               qTileBlockGlobalMemIdx);
      }
#endif

#ifdef DEBUG2
      if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
          (threadIdx.y == 0)) {
        printf("qTileIdx=%d: qTileEnd: %d, qTileGridXYGlobalMemIdx: %d, "
               "qTileBlockGlobalMemIdx: %d\n",
               qTileIdx, qTileEnd, qTileGridXYGlobalMemIdx,
               qTileBlockGlobalMemIdx);
      }
#endif
      //! qTile Loading
      // loops over rows (outer) and columns (inner) of qTile
      const uint qWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
      const uint qWarpTileXEnd = CEIL_DIV(dimHeads, blockDim.x);
      for (uint qWarpTileYIdx = 0; qWarpTileYIdx < qWarpTileYEnd;
           ++qWarpTileYIdx) {
#ifdef DEBUG2
        if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
            (threadIdx.y == 0)) {
          printf("qWarpTileYIdx=%d: qWarpTileYEnd: %d, qWarpTileXEnd: %d\n",
                 qWarpTileYIdx, qWarpTileYEnd, qWarpTileXEnd);
        }
#endif
        for (uint qWarpTileXIdx = 0; qWarpTileXIdx < qWarpTileXEnd;
             ++qWarpTileXIdx) {
          //? qWarpTileIdxes
          //* shared memory:
          const uint qWarpTileThreadSharedMemYIdx =
              blockDim.y * qWarpTileYIdx + threadIdx.y;
          const uint qWarpTileThreadSharedMemXIdx =
              blockDim.x * qWarpTileXIdx + threadIdx.x;
          //* global memory:
          // left upper corner of qTileBlock in Q (global memory)
          const uint qWarpTileBlockGlobalMemIdx =
              qTileBlockGlobalMemIdx + (dimHeads * blockDim.y) * qWarpTileYIdx +
              blockDim.x * qWarpTileXIdx;
          const uint qWarpTileThreadGlobalMemIdx =
              qWarpTileBlockGlobalMemIdx + dimHeads * threadIdx.y + threadIdx.x;

          SMEMARRAY(qTile, dimHeads, qWarpTileThreadSharedMemYIdx,
                    qWarpTileThreadSharedMemXIdx) =
              matQ[qWarpTileThreadGlobalMemIdx];

#ifdef DEBUG3
          if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
              (threadIdx.y == 0)) {
            printf("qTile[%d][%d] = %f\n", qWarpTileThreadSharedMemYIdx,
                   qWarpTileThreadSharedMemXIdx,
                   type2float(qTile[qWarpTileThreadSharedMemYIdx]
                                   [qWarpTileThreadSharedMemXIdx]));
          }
#endif
        }
      }
      __syncthreads(); // TODO: necessary?

      //? flatten the threads to 1D
      const uint flatThreadIdx = blockDim.x * threadIdx.y + threadIdx.x;

      //! init fTileCol to fTileColLast
      // Y: seqLen (or QtileDim), X: 1 (fTileCol has only Y dimension)
      const uint fTileColChunkYEnd =
          CEIL_DIV(QtileDim, blockDim.x * blockDim.y);
      for (uint fTileColChunkYIdx = 0; fTileColChunkYIdx < fTileColChunkYEnd;
           ++fTileColChunkYIdx) {
        //? f idxes
        //* shared memory:
        const uint fThreadSharedMemYIdx =
            flatThreadIdx + blockDim.x * blockDim.y * fTileColChunkYIdx;

        if (fThreadSharedMemYIdx < QtileDim) {
          SMEMVECTOR(fTileCol, fThreadSharedMemYIdx) =
              SMEMVECTOR(fTileColLast, 0);
        }
      }
      __syncthreads();

      // looplevel 2: loop over KVtile blocks along seqLen dim
      //! For causal computation: kvTileIdx <= qTileIdx * gridDim.y +
      //! blockIdx.y
      // other working version: kvTileIdx < kvTileEnd (inefficient due to
      // loading of unnecessary numbers)
      const uint kvTileEnd = qTileIdx * gridDim.y + blockIdx.y + 1;
      for (uint kvTileIdx = 0; kvTileIdx < kvTileEnd; ++kvTileIdx) {

        //* offset in K&V matrix for kTile & vTile (global memory)
        // (k-tile & v-tile have the same BlockGlobalMemIdx, we just change the
        // pointer to the respective memory space to load the k-tile and
        // v-tile).
        const uint kvTileBlockGlobalMemIdx =
            batchHeadGridXGlobalMemIdxQKV + (dimHeads * KVtileDim) * kvTileIdx;

        //* (grid&block) offset X-axis in C = Q*K^T matrix
        // (along sequence dimension) (used for checking causality)
        const uint cTileGridXIdx = KVtileDim * kvTileIdx;
        const uint cTileBlockXIdx = cTileGridXIdx;

        //! kTile & vTile Loading
        // loops over rows (outer) and columns (inner) of kTile & vTile
        const uint kvWarpTileYEnd = CEIL_DIV(KVtileDim, blockDim.y);
        const uint kvWarpTileXEnd = CEIL_DIV(dimHeads, blockDim.x);
        for (uint kvWarpTileYIdx = 0; kvWarpTileYIdx < kvWarpTileYEnd;
             ++kvWarpTileYIdx) {

#ifdef DEBUG2
          if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
              (threadIdx.y == 0)) {
            printf(
                "kvWarpTileYIdx=%d: kvWarpTileYEnd: %d, kvWarpTileXEnd: %d\n",
                kvWarpTileYIdx, kvWarpTileYEnd, kvWarpTileXEnd);
          }
#endif
          for (uint kvWarpTileXIdx = 0; kvWarpTileXIdx < kvWarpTileXEnd;
               ++kvWarpTileXIdx) {
            //? kvWarpTileIdxes for k-tile AND v-tile
            //* shared memory:
            const uint kvWarpTileThreadSharedMemYIdx =
                blockDim.y * kvWarpTileYIdx + threadIdx.y;
            const uint kvWarpTileThreadSharedMemXIdx =
                blockDim.x * kvWarpTileXIdx + threadIdx.x;
            //* global memory:
            // left upper corner of kTileBlock in K (global memory)
            const uint kvWarpTileBlockGlobalMemIdx =
                kvTileBlockGlobalMemIdx +
                (dimHeads * blockDim.y) * kvWarpTileYIdx +
                blockDim.x * kvWarpTileXIdx;
            const uint kvWarpTileThreadGlobalMemIdx =
                kvWarpTileBlockGlobalMemIdx + dimHeads * threadIdx.y +
                threadIdx.x;

            SMEMARRAY(kTile, dimHeads, kvWarpTileThreadSharedMemYIdx,
                      kvWarpTileThreadSharedMemXIdx) =
                matK[kvWarpTileThreadGlobalMemIdx];
            SMEMARRAY(vTile, dimHeads, kvWarpTileThreadSharedMemYIdx,
                      kvWarpTileThreadSharedMemXIdx) =
                matV[kvWarpTileThreadGlobalMemIdx];
          }
        }
        __syncthreads();

        //! construct fTileCol for dTile computation
        // TODO maybe use a parallel scan for optimization (each thread
        // basically does the same computation)
        // fTileCol is the first column of
        // the fgates for the current dTile use all threads along Y (seqLen,
        // qTileDim) dimension to compute the sums of the forgetgates in
        // parallel

        // end idx for the chunkwise loop over fGatePreacts with flattened
        // thread block
        const uint fWarpChunkEnd = CEIL_DIV(QtileDim, blockDim.x * blockDim.y);
        if (kvTileIdx == 0) {
          // we begin at seqLen position 0 in X direction
          // we compute the cumulative sum of the forget gates per row position
          // in the dTile

          // loop chunkwise over the fGatePreacts up to the current qTile
          // position
          const uint fChunkEnd = gridDim.y * qTileIdx + blockIdx.y + 1;
          // TODO optimize by setting the fChunkStartIdx properly
          for (uint fChunkIdx = 0; fChunkIdx < fChunkEnd; ++fChunkIdx) {
            //? f idxes
            // load fChunk for fChunkIdx
            //* (grid&block) offset in f preactivations for fChunk (global
            // memory)
            const uint fChunkGridXYGlobalMemIdx =
                batchHeadGridXGlobalMemIdxIFgate +
                (1 * QtileDim * gridDim.y) * fChunkIdx;
            const uint fChunkBlockGlobalMemIdx =
                fChunkGridXYGlobalMemIdx + (1 * QtileDim) * blockIdx.y;

            //? loading fChunk into shared memory with threadblocks
            for (uint fWarpChunkIdx = 0; fWarpChunkIdx < fWarpChunkEnd;
                 ++fWarpChunkIdx) {
              //? f idxes
              //* shared memory:
              const uint fThreadSharedMemYIdx =
                  flatThreadIdx + blockDim.x * blockDim.y * fWarpChunkIdx;
              //* global memory:
              const uint fThreadGlobalMemIdx =
                  fChunkBlockGlobalMemIdx + fThreadSharedMemYIdx;

              if (fThreadSharedMemYIdx < QtileDim) {
                // TODO apply log_simoid to the loaded fGatePreacts
                SMEMVECTOR(fChunk, fThreadSharedMemYIdx) =
                    fGatePreact[fThreadGlobalMemIdx];
                //   SMEMVECTOR(fChunk, fThreadSharedMemYIdx) =
                //       logsigmoid_g(fGatePreact[fThreadGlobalMemIdx]);
              }
            }
            __syncthreads();

            //? sum up fgates for dTile
            // the very first forgetgate index must be f_2
            for (uint fWarpChunkIdx = 0; fWarpChunkIdx < fWarpChunkEnd;
                 ++fWarpChunkIdx) {
              //? f idxes
              //* shared memory:
              const uint fThreadSharedMemYIdx =
                  flatThreadIdx + blockDim.x * blockDim.y * fWarpChunkIdx;

              //? d idxes
              //* (thread) offset Y-axis (seqLen, QTileDim) of dTile (global)
              const uint dTileThreadYIdx =
                  cTileBlockYIdx + fThreadSharedMemYIdx;

              float f_acc;
              if (fThreadSharedMemYIdx < QtileDim) {
                // init forget gate accumulator
                if (fWarpChunkIdx == 0) {
                  f_acc = 0.0f;
                } else {
                  f_acc = SMEMVECTOR(fTileCol, fThreadSharedMemYIdx);
                }

                // start the sum at the second index (corresponds to f_2)
                uint startIdx = 0;
                if (fChunkIdx == 0) {
                  startIdx = 1;
                }
                for (uint i = startIdx; i < QtileDim; ++i) {
                  //? f idxes
                  // fSumIdx corresponds to the current fGatePreact index
                  // (starting from 0) i.e. for f_2: fSumIdx = 1, for f_3:
                  // fSumIdx = 2, ...
                  const uint fSumIdx = fChunkIdx * QtileDim + i;
                  if (fSumIdx > dTileThreadYIdx) {
                    break;
                  }
                  f_acc = add_g(f_acc, type2float(SMEMVECTOR(fChunk, i)));
                }
                SMEMVECTOR(fTileCol, fThreadSharedMemYIdx) = f_acc;
              }
            }
            __syncthreads();
          }

          // todo sync grid?
        }
        // else: do nothing
        // we are within the sequence at position > kvTileDim * kvTileIdx
        // we can just use the fTileCol from the previous iteration and keep
        // subtracting we only need to update the fTileCol for the next
        // kvTileIdx at the end of the current kvTileIdx iteration

        //! iChunk&fChunk Loading
        //* (grid&block) offset in i&f preactivations for i and f chunk (global
        // memory)
        // every thread block loads the same i&f preactivations
        const uint ifChunkBlockXYGlobalMemIdx =
            batchHeadGridXGlobalMemIdxIFgate + (1 * KVtileDim) * kvTileIdx;

        // Y: seqLen (or KVtileDim), X: 1
        // we only load the fGatePreacts for the current kvTileIdx
        const uint iChunkChunkYEnd =
            CEIL_DIV(KVtileDim, blockDim.x * blockDim.y);
        for (uint iChunkYIdx = 0; iChunkYIdx < iChunkChunkYEnd; ++iChunkYIdx) {
          //? i idxes
          //* shared memory:
          const uint ifThreadSharedMemYIdx =
              flatThreadIdx + blockDim.x * blockDim.y * iChunkYIdx;
          //* global memory:
          const uint ifChunkThreadGlobalMemIdx =
              ifChunkBlockXYGlobalMemIdx + ifThreadSharedMemYIdx;

          if (ifThreadSharedMemYIdx < KVtileDim) {
            SMEMVECTOR(iChunk, ifThreadSharedMemYIdx) =
                iGatePreact[ifChunkThreadGlobalMemIdx];
            SMEMVECTOR(fChunk, ifThreadSharedMemYIdx) =
                fGatePreact[ifChunkThreadGlobalMemIdx];
          }
        }
        __syncthreads();

        //! construct dTile
        // go over tile from left to right in kvTileDim dimension, subtract
        // fgates again, add igates while going over it compute the max state
        // for the dTile (keep max in register and copy to shared memory at the
        // end)

        for (uint fWarpChunkIdx = 0; fWarpChunkIdx < fWarpChunkEnd;
             ++fWarpChunkIdx) {
          //? f idxes
          //* shared memory:
          const uint fThreadSharedMemYIdx =
              flatThreadIdx + blockDim.x * blockDim.y * fWarpChunkIdx;

          //? d idxes
          //* (thread) [local] offset Y-axis (seqLen, QTileDim) of dTile
          const uint dTileLocalThreadYIdx = fThreadSharedMemYIdx;
          //* (thread) [global] offset Y-axis (seqLen, QTileDim) of dTile
          const uint dTileThreadYIdx = cTileBlockYIdx + dTileLocalThreadYIdx;

          if (fThreadSharedMemYIdx < QtileDim) {
            //* (thread) [local] offset X-axis (KVtileDim) of dTile
            const uint dTileLocalThreadXIdx = kvTileIdx;

            float f_acc_subtractfrom =
                SMEMVECTOR(fTileCol, fThreadSharedMemYIdx);
            float d_max = f_acc_subtractfrom;
            float d_val = 0.0f;
            for (uint i = 0; i < KVtileDim; ++i) {
              //* (thread) [global] offset X-axis (KVtileDim) of dTile
              const uint dTileThreadXIdx = cTileBlockXIdx + i;

              // f gate only
              if (dTileThreadXIdx == dTileThreadYIdx) {
                // set to 0
                f_acc_subtractfrom = 0.0f;
              } else if (dTileThreadXIdx == 0) {
                // first column of dTile
                // no change to f_acc_subtractfrom
              } else if (dTileThreadXIdx > dTileThreadYIdx) {
                // set to negative infinity
                f_acc_subtractfrom = -HIP_INF_F;
              } else {
                // dTileThreadXIdx < dTileThreadYIdx
                // subtract f gate
                f_acc_subtractfrom = sub_g(f_acc_subtractfrom,
                                           type2float(SMEMVECTOR(fChunk, i)));
              }
              // d_val;
              d_val = f_acc_subtractfrom; // TODO change to this
                                          // add_g(f_acc_subtractfrom,
                                          // type2float(SMEMVECTOR(iChunk, i)));

              // max state
              d_max = max_g(d_max, d_val);

              // write d_val into to dTile shared memory
              SMEMARRAY(dTile, KVtileDim, dTileLocalThreadYIdx,
                        dTileLocalThreadXIdx) = float2type<scalar_t>(d_val);
            }
            // save max state of dTile in shared memory
            SMEMVECTOR(mChunk, dTileLocalThreadYIdx) = d_max;
            // save last f_acc_subtractfrom in fTileCol for next kvTileIdx
            SMEMVECTOR(fTileCol, fThreadSharedMemYIdx) = f_acc_subtractfrom;
          }
        }
        __syncthreads();

        //! compute C = (Q x K^T) * dTile, i.e. fill cTile
        // (QtileDim,KVtileDim) = (QtileDim,dimHeads) x (dimHeads,KVtileDim)
        // loops over cTile rows (outer) and columns (inner)
        const uint cWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
        const uint cWarpTileXEnd = CEIL_DIV(KVtileDim, blockDim.x);
        for (uint cWarpTileYIdx = 0; cWarpTileYIdx < cWarpTileYEnd;
             ++cWarpTileYIdx) {
          //* (thread) offset Y-axis in C = Q*K^T
          const uint cTileThreadYIdx =
              cTileBlockYIdx + blockDim.y * cWarpTileYIdx + threadIdx.y;

          for (uint cWarpTileXIdx = 0; cWarpTileXIdx < cWarpTileXEnd;
               ++cWarpTileXIdx) {
            //? cTileIdxes
            //* shared memory:
            const uint cWarpTileThreadSharedMemYIdx =
                blockDim.y * cWarpTileYIdx + threadIdx.y;
            const uint cWarpTileThreadSharedMemXIdx =
                blockDim.x * cWarpTileXIdx + threadIdx.x;

            //* (thread) offset X-axis in C = Q*K^T
            const uint cTileThreadXIdx =
                cTileBlockXIdx + blockDim.x * cWarpTileXIdx + threadIdx.x;

            // scalar_t qk_acc = dscalar_zero<scalar_t>();
            float qk_acc = 0.0f;
            //! check for causality here
            // compute only the lower triangle (below main diagonal) of C =
            // Q*K^T
            if (cTileThreadXIdx <= cTileThreadYIdx) {
              for (uint i = 0; i < dimHeads; ++i) {
                qk_acc = add_g(
                    qk_acc, type2float(mul_g(
                                SMEMARRAY(qTile, dimHeads,
                                          cWarpTileThreadSharedMemYIdx, i),
                                SMEMARRAY(kTile, dimHeads,
                                          cWarpTileThreadSharedMemXIdx, i))));
#ifdef DEBUG4
                if ((blockIdx.x == 0) && (blockIdx.y == 0) &&
                    (threadIdx.x == 0) && (threadIdx.y == 3) &&
                    (cWarpTileXIdx == 0) && (kvTileIdx == 0) &&
                    (i == dimHeads - 1)) {
                  printf("qTIdx=%d|kvTIdx=%d: qTile[%d][%d] = %f\n", qTileIdx,
                         kvTileIdx, cWarpTileThreadSharedMemYIdx, i,
                         type2float(qTile[cWarpTileThreadSharedMemYIdx][i]));
                  printf("qTIdx=%d|kvTIdx=%d: kTile[%d][%d] = %f\n", qTileIdx,
                         kvTileIdx, cWarpTileThreadSharedMemXIdx, i,
                         type2float(kTile[cWarpTileThreadSharedMemXIdx][i]));
                  printf("qTIdx=%d|kvTIdx=%d: cTile[%d][%d](%d) = %f\n",
                         qTileIdx, kvTileIdx, cWarpTileThreadSharedMemYIdx,
                         cWarpTileThreadSharedMemXIdx, i, type2float(qk_acc));
                }
#endif
              }
            }
            SMEMARRAY(cTile, KVtileDim, cWarpTileThreadSharedMemYIdx,
                      cWarpTileThreadSharedMemXIdx) =
                float2type<scalar_t>(qk_acc);
            __syncthreads();
          }
        }

        //! compute "raw normalizer" l as rowsum of cTile
        //! compute normalizer n as max(abs(l),exp(-m))
        // TODO implement this

        // TODO reweight the previous H tile

        //! DEBUG only: write dTile to global memory
        // TODO from here

        //! compute H += C * V, i.e. fill hTile
        //! accumulate KVtiles to hTile
        // (QtileDim,dimHeads) = (QtileDim,KVtileDim) x (KVtileDim,dimHeads)
        // loops over hTile rows (outer) and columns (inner)
        const uint hWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
        const uint hWarpTileXEnd = CEIL_DIV(dimHeads, blockDim.x);
        for (uint hWarpTileYIdx = 0; hWarpTileYIdx < hWarpTileYEnd;
             ++hWarpTileYIdx) {
          for (uint hWarpTileXIdx = 0; hWarpTileXIdx < hWarpTileXEnd;
               ++hWarpTileXIdx) {

            //? cTileIdxes
            //* shared memory:
            const uint hWarpTileThreadSharedMemYIdx =
                blockDim.y * hWarpTileYIdx + threadIdx.y;
            const uint hWarpTileThreadSharedMemXIdx =
                blockDim.x * hWarpTileXIdx + threadIdx.x;

            // scalar_t sv_acc = dscalar_zero<scalar_t>();
            float sv_acc = 0.0f;
            for (uint i = 0; i < KVtileDim; ++i) {
              sv_acc = add_g(
                  sv_acc,
                  type2float(mul_g(SMEMARRAY(cTile, KVtileDim,
                                             hWarpTileThreadSharedMemYIdx, i),
                                   SMEMARRAY(vTile, dimHeads, i,
                                             hWarpTileThreadSharedMemXIdx))));
            }
            // accumulate over all KVtiles
            if (kvTileIdx == 0) {
              // we need to clear the hTile in first iteration
              SMEMARRAY(hTile, dimHeads, hWarpTileThreadSharedMemYIdx,
                        hWarpTileThreadSharedMemXIdx) =
                  float2type<scalar_t>(sv_acc);
            } else {
              SMEMARRAY(hTile, dimHeads, hWarpTileThreadSharedMemYIdx,
                        hWarpTileThreadSharedMemXIdx) =
                  add_g(SMEMARRAY(hTile, dimHeads, hWarpTileThreadSharedMemYIdx,
                                  hWarpTileThreadSharedMemXIdx),
                        float2type<scalar_t>(sv_acc));
            }
            __syncthreads();
          }
        }
      }

      // TODO sync all blocks here, necessary? The loop above has different
      // number of iterations for each block
      // gridGroup.sync();

      //! write hTile to global memory (has the same memory index as qTile)
      // loops over hTile rows (outer) and columns (inner)
      const uint hWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
      const uint hWarpTileXEnd = CEIL_DIV(dimHeads, blockDim.x);
      for (uint hWarpTileYIdx = 0; hWarpTileYIdx < hWarpTileYEnd;
           ++hWarpTileYIdx) {
        for (uint hWarpTileXIdx = 0; hWarpTileXIdx < hWarpTileXEnd;
             ++hWarpTileXIdx) {

          //? cTileIdxes
          //* shared memory:
          const uint hWarpTileThreadSharedMemYIdx =
              blockDim.y * hWarpTileYIdx + threadIdx.y;
          const uint hWarpTileThreadSharedMemXIdx =
              blockDim.x * hWarpTileXIdx + threadIdx.x;
          //* global memory:
          // left upper corner of cWarpTileBlock in C (global memory)
          const uint hWarpTileBlockGlobalMemIdx =
              qTileBlockGlobalMemIdx + (dimHeads * blockDim.y) * hWarpTileYIdx +
              blockDim.x * hWarpTileXIdx;
          const uint hWarpTileThreadGlobalMemIdx =
              hWarpTileBlockGlobalMemIdx + dimHeads * threadIdx.y + threadIdx.x;

          matH[hWarpTileThreadGlobalMemIdx] =
              SMEMARRAY(hTile, dimHeads, hWarpTileThreadSharedMemYIdx,
                        hWarpTileThreadSharedMemXIdx);
        }
      }
      __syncthreads();
    }
  }
}

template <typename scalar_t>
void kernel_dispatchers::vlstm_fw_dispatch(
    scalar_t *matH, scalar_t *matC, scalar_t *matQ, scalar_t *matK,
    scalar_t *matV, scalar_t *iGatePreact, scalar_t *fGatePreact, int batchSize,
    int numHeads, int seqLen, int dimHeads) {
  printf("B: %d, NH: %d, S: %d, DH: %d\n", batchSize, numHeads, seqLen,
         dimHeads);
  const int TblockDim = TBLOCK_DIM; // matmul blockdim
  const int QtileDim = QTILE_DIM;   // blockdim for Q along seqLen dim
  const int KVtileDim = KVTILE_DIM; // blockdim for K&V along seqLen dim

  // kernel asserts
  if ((seqLen % QtileDim != 0) || (seqLen % KVtileDim != 0)) {
    printf("seqLen must be divisible by QblockDim and KVblockDim\n");
  }

  // determine the number of blocks and threads
  const dim3 blockDims(TblockDim, TblockDim);

  // TODO: determine gridDims
  // Note @mbeck: should be dynamically allocated.
  // At first parallelize across batchSize and numHeads.
  // If more streaming multiprocessors available, parallelize across seqLen.
  //! NOTE: for now we only parallelize across batchSize and numHeads
  // TODO Need to dynamically check how many blocks we can launch
  // TODO add check if batchSize*numHeads exceeds max gridDim.x

  const dim3 gridDims(batchSize * numHeads, 2);
  // const dim3 gridDims(1, 1);

  //! calculate dynamic shared memory size
  // TODO understand how memory padding works!
  // Why at innermost dim? Because memory is organized consecutively
  // we are storing the following tiles in shared memory:
  // - Input tiles: qTile, vTile, kTile -> (QtileDim, dimHeads +
  // SHARED_MEM_PADDING)
  // TODO from here add input & forgetgate tiles
  // - Intermediate result tile: cTile, dTile -> (QtileDim, KVtileDim +
  // SHARED_MEM_PADDING)
  // - Output tile: hTile -> (QtileDim, dimHeads + SHARED_MEM_PADDING)

  const uint qkvhTileSharedMemSize =
      sizeof(scalar_t) * QtileDim * (dimHeads + SHARED_MEM_PADDING);
  const uint cdTileSharedMemSize =
      sizeof(scalar_t) * QtileDim * (KVtileDim + SHARED_MEM_PADDING);

  // See here:
  // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#device-memory-accesses
  // the idea of the padding is that every number is stored in a different
  // memory bank this should help to avoid bank conflicts as many threads need
  // to access the same input and forget gate values at the same time for the
  // gate matrix computation
  // TODO check if this is really helping!
  const uint iChunkSharedMemSize =
      sizeof(scalar_t) * KVtileDim * (1 + SHARED_MEM_PADDING);
  const uint fChunkSharedMemSize =
      sizeof(scalar_t) * QtileDim * (1 + SHARED_MEM_PADDING);

  // we keep these as float as it acts as accumulator
  const uint fTileColSharedMemSize =
      sizeof(float) * QtileDim * (1 + SHARED_MEM_PADDING);
  const uint fTileColLastSharedMemSize =
      sizeof(float) * 1 * (1 + SHARED_MEM_PADDING);

  const uint mChunkSharedMemSize =
      sizeof(scalar_t) * QtileDim * (1 + SHARED_MEM_PADDING);
  const uint lChunkSharedMemSize =
      sizeof(scalar_t) * QtileDim * (1 + SHARED_MEM_PADDING);

  // Input/Output tiles: 4x for qTile, vTile, kTile, hTile
  // Intermediate tiles: 2x for cTile, dTile
  // Intermediate tiles: 2x for mChunk, lChunk
  const uint sharedMemorySize =
      4 * qkvhTileSharedMemSize + 2 * cdTileSharedMemSize +
      iChunkSharedMemSize + fChunkSharedMemSize + fTileColSharedMemSize +
      2 * mChunkSharedMemSize + 2 * lChunkSharedMemSize +
      fTileColLastSharedMemSize;

  printf("blocksxy: %d-%d, threadsxy: %d-%d, shared_mem in bytes: %d\n",
         gridDims.x, gridDims.y, blockDims.x, blockDims.y, sharedMemorySize);
  // hipSetDevice(0);

  hipStream_t stream;
  hipStreamCreate(&stream);

  auto kernel = kernels::vlstm_fw<scalar_t, TblockDim, QtileDim, KVtileDim>;
  hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributePreferredSharedMemoryCarveout,
                       cudaSharedmemCarveoutMaxShared);
  hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize,
                       sharedMemorySize);
  // define void* pointers to the kernel arguments
  void *kernelArgs[] = {
      (void *)&matH,        (void *)&matC,      (void *)&matQ,
      (void *)&matK,        (void *)&matV,      (void *)&iGatePreact,
      (void *)&fGatePreact, (void *)&batchSize, (void *)&numHeads,
      (void *)&seqLen,      (void *)&dimHeads};

  hipLaunchCooperativeKernel((void *)kernel, gridDims, blockDims, kernelArgs,
                              sharedMemorySize, stream);

  gpuErrchk(hipPeekAtLastError());

  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);
  gpuErrchk(hipDeviceSynchronize());

  // gpuErrchk(hipPeekAtLastError());
  // gpuErrchk(hipDeviceSynchronize());
}

// this is needed to make sure that the compiler instantiates the template
template void kernel_dispatchers::vlstm_fw_dispatch<__hip_bfloat16>(
    __hip_bfloat16 *matH, __hip_bfloat16 *matC, __hip_bfloat16 *matQ,
    __hip_bfloat16 *matK, __hip_bfloat16 *matV, __hip_bfloat16 *iGatePreact,
    __hip_bfloat16 *fGatePreact, int batchSize, int numHeads, int seqLen,
    int dimHeads);
template void kernel_dispatchers::vlstm_fw_dispatch<__half>(
    __half *matH, __half *matC, __half *matQ, __half *matK, __half *matV,
    __half *iGatePreact, __half *fGatePreact, int batchSize, int numHeads,
    int seqLen, int dimHeads);
template void kernel_dispatchers::vlstm_fw_dispatch<float>(
    float *matH, float *matC, float *matQ, float *matK, float *matV,
    float *iGatePreact, float *fGatePreact, int batchSize, int numHeads,
    int seqLen, int dimHeads);

} // namespace vlstm