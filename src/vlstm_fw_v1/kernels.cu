#include "hip/hip_runtime.h"
// Copyright JKU Linz 2023
// Author: Maximilian Beck

#include <hip/hip_cooperative_groups.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

#include "../util/cuda_errorcheck.h"
#include "../util/inline_ops.cuh"
#include "../util/inline_print.cuh"
#include "../util/support.h"
#include "kernel_dispatchers.h"

#define CEIL_DIV(a, b) (((a) + (b)-1) / (b))
#define MAX(a, b) (((a) > (b)) ? (a) : (b))
#define MIN(a, b) (((a) < (b)) ? (a) : (b))

namespace vlstm {

namespace cg = cooperative_groups;

namespace kernels {

template <typename scalar_t, int TblockDim, int QblockDim, int KVblockDim>
__global__ void vlstm_fw(scalar_t *matH, scalar_t *matQ, scalar_t *matK,
                         scalar_t *matV, scalar_t *iGatePreact,
                         scalar_t *fGatePreact, int batchSize, int numHeads,
                         int seqLen, int dimHeads);

} // namespace kernels

////////////////////////////////////////////////////////////////////////////////////////

#define TBLOCK_DIM 4 // TblockDim: corresponds to BLOCK_DIM in matmul
#define KVTILE_DIM 8 // KVtileDim: TileDim for K&V along seqLen dim
// QTILE_DIM must be divisible by KVTILE_DIM and TBLOCK_DIM
#define QTILE_DIM 8 // QtileDim: TileDim for Q along seqLen dim

// shared memory must be aligned: depends on scalar_t (multiples of 4 should be
// fine for bf16, fp16 and fp32)
#define SHARED_MEM_PADDING 8 // SHARED_MEM_PADDING: padding for shared memory

// SMEMARRAY: access shared memory array (2D)
#define SMEMARRAY(array, stride, row, col)                                     \
  array[(row) * (stride + SHARED_MEM_PADDING) + (col)]
// SMEMVECTOR: access shared memory vector (1D)
#define SMEMVECTOR(array, idx) array[(idx) * (1 + SHARED_MEM_PADDING)]

#define DEBUG 1
// #define DEBUG2 1
// #define DEBUG3 1
// #define DEBUG4 1
// #define DEBUG5 1

/**
Conventions:
// TODO add conventions

*/

/* vLSTM Forward Kernel v0 */

template <typename scalar_t, int TblockDim, int QtileDim, int KVtileDim>
__global__ void
kernels::vlstm_fw(scalar_t *matH, scalar_t *matQ, scalar_t *matK,
                  scalar_t *matV, scalar_t *iGatePreact, scalar_t *fGatePreact,
                  int batchSize, int numHeads, int seqLen, int dimHeads) {
  // int tIdx = threadIdx.x + blockDim.x * threadIdx.y;
#ifdef DEBUG
  if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
      (threadIdx.y == 0)) {
    printf("In Kernel: gdim.x: %d, gdim.y: %d, gdim.z: %d, bdim.x: %d, bdim.y: "
           "%d\n",
           gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y);
  }
#endif

#ifdef DEBUG
  if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
      (threadIdx.y == 0)) {
    printf("In Kernel: QtileDim: %d, KVtileDim: %d, TblockDim:%d\n", QtileDim,
           KVtileDim, TblockDim);
  }
#endif
  cg::grid_group gridGroup = cg::this_grid();

  //! Shared Memory aka SRAM
  // the data in this shared memory is shared across all threads in a thread
  // block
  extern __shared__ float sbuf[]; // declare it as float and redefine it later

  //? for inputs
  // Note: keep in mind the memory is defined in a contiguous region.
  // One pointer has the full memory space until the next point is defined.
  // Therefore to read the size of a single shared memory array you need to
  // have a look at the offset for the next array.

  // qtile (QtileDim x dimHeads) in shared memory (padding for alignment)
  scalar_t *qTile = (scalar_t *)sbuf;
  // kTile and vTile (KVtileDim x dimHeads) in shared memory
  scalar_t *kTile =
      (scalar_t *)&qTile[QtileDim * (dimHeads + SHARED_MEM_PADDING)];
  scalar_t *vTile =
      (scalar_t *)&kTile[KVtileDim * (dimHeads + SHARED_MEM_PADDING)];

  //? for intermediate results
  // init cTile (QtileDim x KVTileDim) in shared memory for intermediate
  // result of QK^T
  scalar_t *cTile =
      (scalar_t *)&vTile[KVtileDim * (dimHeads + SHARED_MEM_PADDING)];
  // init result hTile (QTileDim x dimHeads) in shared memory
  scalar_t *hTile =
      (scalar_t *)&cTile[QtileDim * (KVtileDim + SHARED_MEM_PADDING)];
  // init dTile (QTileDim x KVTileDim) in shared memory for forget and input
  // gate matrix
  scalar_t *dTile =
      (scalar_t *)&hTile[QtileDim * (dimHeads + SHARED_MEM_PADDING)];

  //? for input and forget gate
  // init iChunk (KVTileDim x 1) in shared memory for input gate
  scalar_t *iChunk =
      (scalar_t *)&dTile[QtileDim * (dimHeads + SHARED_MEM_PADDING)];
  // init fChunk (QTileDim x 1) in shared memory for forget gate
  scalar_t *fChunk = (scalar_t *)&iChunk[KVtileDim * (1 + SHARED_MEM_PADDING)];
  // init fTileCol (QTileDim x 1) in shared memory for forget gate (first column
  // of the QtileDim x KVtileDim dTile)
  float *fTileCol = (float *)&fChunk[QtileDim * (1 + SHARED_MEM_PADDING)];
  // init fTileColLast (1 x 1) in shared memory for forget gate (last row value
  // of fTileCol)
  float *fTileColLast = (float *)&fTileCol[QtileDim * (1 + SHARED_MEM_PADDING)];

  //! PARALLELIZE ALONG BATCHSIZE * NUMHEADS (gridDim.x)
  const uint batchHeadStepQKV = seqLen * dimHeads;
  const uint batchHeadStepIFgate = seqLen * 1;
  const uint numBatchHeads = batchSize * numHeads;
  // End for looplevel 0:
  const uint batchHeadEnd = CEIL_DIV(numBatchHeads, gridDim.x);
  // looplevel 0: loop over batches and heads
  for (uint batchHeadIdx = 0; batchHeadIdx < batchHeadEnd; ++batchHeadIdx) {

    uint batchHeadGridXGlobalMemIdxQKV =
        (batchHeadStepQKV * gridDim.x) * batchHeadIdx +
        (batchHeadStepQKV)*blockIdx.x;

    uint batchHeadGridXGlobalMemIdxIFgate =
        (batchHeadStepIFgate * gridDim.x) * batchHeadIdx +
        (batchHeadStepIFgate)*blockIdx.x;

#ifdef DEBUG5
    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
      printf("B<%d,%d> batchHeadIdx: %d, batchHeadEnd: %d, "
             "batchHeadGridXGlobalMemIdxQKV: "
             "%d\n",
             blockIdx.x, blockIdx.y, batchHeadIdx, batchHeadEnd,
             batchHeadGridXGlobalMemIdxQKV);
    }
#endif
    SMEMVECTOR(fTileColLast, 0) =
        float2type<float>(0.0f); // could also just write 0.0f

    //! PARALLELIZE ALONG SEQLEN (gridDim.y)
    // Ends for looplevel 1:
    const uint qTileEnd = CEIL_DIV(seqLen, QtileDim * gridDim.y);
    // looplevel 1: loop over Qtile blocks along seqLen dim
    for (uint qTileIdx = 0; qTileIdx < qTileEnd; ++qTileIdx) {

      //* qTile Global Memory Index
      // (grid&block) offset in Q matrix for qTile (global memory)
      const uint qTileGridXYGlobalMemIdx =
          batchHeadGridXGlobalMemIdxQKV +
          (dimHeads * QtileDim * gridDim.y) * qTileIdx;
      const uint qTileBlockGlobalMemIdx =
          qTileGridXYGlobalMemIdx + (dimHeads * QtileDim) * blockIdx.y;

      //* (grid&block) offset in f preactivations for fChunk (global memory)
      const uint fChunkGridXYGlobalMemIdx =
          batchHeadGridXGlobalMemIdxIFgate +
          (1 * QtileDim * gridDim.y) * qTileIdx;
      const uint fChunkBlockGlobalMemIdx =
          fChunkGridXYGlobalMemIdx + (1 * QtileDim) * blockIdx.y;

      //* cTile Global Memory Index (virtual, as never materialized fully)
      // (grid&block) offset Y-axis in C = Q*K^T matrix (along sequence
      // dimension) (used for checking causality)
      const uint cTileGridYIdx = QtileDim * gridDim.y * qTileIdx;
      const uint cTileBlockYIdx = cTileGridYIdx + QtileDim * blockIdx.y;

#ifdef DEBUG5
      if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        printf("B<%d,%d> qTileIdx: %d, qTileEnd: %d, "
               "qTileBlockGlobalMemIdx: "
               "%d, \n",
               blockIdx.x, blockIdx.y, qTileIdx, qTileEnd,
               qTileBlockGlobalMemIdx);
      }
#endif

#ifdef DEBUG2
      if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
          (threadIdx.y == 0)) {
        printf("qTileIdx=%d: qTileEnd: %d, qTileGridXYGlobalMemIdx: %d, "
               "qTileBlockGlobalMemIdx: %d\n",
               qTileIdx, qTileEnd, qTileGridXYGlobalMemIdx,
               qTileBlockGlobalMemIdx);
      }
#endif
      //! qTile Loading
      // loops over rows (outer) and columns (inner) of qTile
      const uint qWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
      const uint qWarpTileXEnd = CEIL_DIV(dimHeads, blockDim.x);
      for (uint qWarpTileYIdx = 0; qWarpTileYIdx < qWarpTileYEnd;
           ++qWarpTileYIdx) {
#ifdef DEBUG2
        if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
            (threadIdx.y == 0)) {
          printf("qWarpTileYIdx=%d: qWarpTileYEnd: %d, qWarpTileXEnd: %d\n",
                 qWarpTileYIdx, qWarpTileYEnd, qWarpTileXEnd);
        }
#endif
        for (uint qWarpTileXIdx = 0; qWarpTileXIdx < qWarpTileXEnd;
             ++qWarpTileXIdx) {
          //? qWarpTileIdxes
          //* shared memory:
          const uint qWarpTileThreadSharedMemYIdx =
              blockDim.y * qWarpTileYIdx + threadIdx.y;
          const uint qWarpTileThreadSharedMemXIdx =
              blockDim.x * qWarpTileXIdx + threadIdx.x;
          //* global memory:
          // left upper corner of qTileBlock in Q (global memory)
          const uint qWarpTileBlockGlobalMemIdx =
              qTileBlockGlobalMemIdx + (dimHeads * blockDim.y) * qWarpTileYIdx +
              blockDim.x * qWarpTileXIdx;
          const uint qWarpTileThreadGlobalMemIdx =
              qWarpTileBlockGlobalMemIdx + dimHeads * threadIdx.y + threadIdx.x;

          SMEMARRAY(qTile, dimHeads, qWarpTileThreadSharedMemYIdx,
                    qWarpTileThreadSharedMemXIdx) =
              matQ[qWarpTileThreadGlobalMemIdx];

#ifdef DEBUG3
          if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
              (threadIdx.y == 0)) {
            printf("qTile[%d][%d] = %f\n", qWarpTileThreadSharedMemYIdx,
                   qWarpTileThreadSharedMemXIdx,
                   type2float(qTile[qWarpTileThreadSharedMemYIdx]
                                   [qWarpTileThreadSharedMemXIdx]));
          }
#endif
        }
      }
      __syncthreads(); // TODO: necessary?

      //? flatten the threads to 1D
      const uint flatThreadIdx = blockDim.x * threadIdx.y + threadIdx.x;

      //! init fTileCol to fTileColLast
      //! fChunk Loading
      // Y: seqLen (or QtileDim), X: 1 (fTileCol has only Y dimension)
      const uint fTileColChunkYEnd =
          CEIL_DIV(QtileDim, blockDim.x * blockDim.y);
      for (uint fTileColChunkYIdx = 0; fTileColChunkYIdx < fTileColChunkYEnd;
           ++fTileColChunkYIdx) {
        //? f idxes
        //* shared memory:
        const uint fThreadSharedMemYIdx =
            flatThreadIdx + blockDim.x * blockDim.y * fTileColChunkYIdx;
        //* global memory:
        const uint fThreadGlobalMemIdx =
            fChunkBlockGlobalMemIdx + fThreadSharedMemYIdx;

        if (fThreadSharedMemYIdx < QtileDim) {
          SMEMVECTOR(fTileCol, fThreadSharedMemYIdx) =
              SMEMVECTOR(fTileColLast, 0);

          SMEMVECTOR(fChunk, fThreadSharedMemYIdx) =
              fGatePreact[fThreadGlobalMemIdx];
        }
      }
      __syncthreads();

      // looplevel 2: loop over KVtile blocks along seqLen dim
      //! For causal computation: kvTileIdx <= qTileIdx * gridDim.y +
      //! blockIdx.y
      // other working version: kvTileIdx < kvTileEnd (inefficient due to
      // loading of unnecessary numbers)
      const uint kvTileEnd = qTileIdx * gridDim.y + blockIdx.y + 1;
      for (uint kvTileIdx = 0; kvTileIdx < kvTileEnd; ++kvTileIdx) {

        //* offset in K&V matrix for kTile & vTile (global memory)
        // (k-tile & v-tile have the same BlockGlobalMemIdx, we just change the
        // pointer to the respective memory space to load the k-tile and
        // v-tile).
        const uint kvTileBlockGlobalMemIdx =
            batchHeadGridXGlobalMemIdxQKV + (dimHeads * KVtileDim) * kvTileIdx;

        //* (grid&block) offset X-axis in C = Q*K^T matrix
        // (along sequence dimension) (used for checking causality)
        const uint cTileGridXIdx = KVtileDim * kvTileIdx;
        const uint cTileBlockXIdx = cTileGridXIdx;

        //* (grid&block) offset in i preactivations for iChunk (global memory)
        const uint iChunkGridXYGlobalMemIdx =
            batchHeadGridXGlobalMemIdxIFgate +
            (1 * KVtileDim * gridDim.y) * kvTileIdx;
        const uint iChunkBlockGlobalMemIdx =
            iChunkGridXYGlobalMemIdx + (1 * KVtileDim) * blockIdx.y;

        //! kTile & vTile Loading
        // loops over rows (outer) and columns (inner) of kTile & vTile
        const uint kvWarpTileYEnd = CEIL_DIV(KVtileDim, blockDim.y);
        const uint kvWarpTileXEnd = CEIL_DIV(dimHeads, blockDim.x);
        for (uint kvWarpTileYIdx = 0; kvWarpTileYIdx < kvWarpTileYEnd;
             ++kvWarpTileYIdx) {

#ifdef DEBUG2
          if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
              (threadIdx.y == 0)) {
            printf(
                "kvWarpTileYIdx=%d: kvWarpTileYEnd: %d, kvWarpTileXEnd: %d\n",
                kvWarpTileYIdx, kvWarpTileYEnd, kvWarpTileXEnd);
          }
#endif
          for (uint kvWarpTileXIdx = 0; kvWarpTileXIdx < kvWarpTileXEnd;
               ++kvWarpTileXIdx) {
            //? kvWarpTileIdxes for k-tile AND v-tile
            //* shared memory:
            const uint kvWarpTileThreadSharedMemYIdx =
                blockDim.y * kvWarpTileYIdx + threadIdx.y;
            const uint kvWarpTileThreadSharedMemXIdx =
                blockDim.x * kvWarpTileXIdx + threadIdx.x;
            //* global memory:
            // left upper corner of kTileBlock in K (global memory)
            const uint kvWarpTileBlockGlobalMemIdx =
                kvTileBlockGlobalMemIdx +
                (dimHeads * blockDim.y) * kvWarpTileYIdx +
                blockDim.x * kvWarpTileXIdx;
            const uint kvWarpTileThreadGlobalMemIdx =
                kvWarpTileBlockGlobalMemIdx + dimHeads * threadIdx.y +
                threadIdx.x;

            SMEMARRAY(kTile, dimHeads, kvWarpTileThreadSharedMemYIdx,
                      kvWarpTileThreadSharedMemXIdx) =
                matK[kvWarpTileThreadGlobalMemIdx];
            SMEMARRAY(vTile, dimHeads, kvWarpTileThreadSharedMemYIdx,
                      kvWarpTileThreadSharedMemXIdx) =
                matV[kvWarpTileThreadGlobalMemIdx];
          }
        }
        __syncthreads();

        //! iChunk Loading
        // Y: seqLen (or KVtileDim), X: 1
        const uint iChunkChunkYEnd =
            CEIL_DIV(KVtileDim, blockDim.x * blockDim.y);
        for (uint iChunkYIdx = 0; iChunkYIdx < iChunkChunkYEnd; ++iChunkYIdx) {
          //? i idxes
          //* shared memory:
          const uint iThreadSharedMemYIdx =
              flatThreadIdx + blockDim.x * blockDim.y * iChunkYIdx;
          //* global memory:
          const uint iThreadGlobalMemIdx =
              iChunkBlockGlobalMemIdx + iThreadSharedMemYIdx;

          if (iThreadSharedMemYIdx < KVtileDim) {
            SMEMVECTOR(iChunk, iThreadSharedMemYIdx) =
                iGatePreact[iThreadGlobalMemIdx];
          }
        }
        __syncthreads();

        //! construct dTile
        // use all threads along Y (seqLen) dimension

        //! compute C = Q x K^T, i.e. fill cTile
        // (QtileDim,KVtileDim) = (QtileDim,dimHeads) x (dimHeads,KVtileDim)
        // loops over cTile rows (outer) and columns (inner)
        const uint cWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
        const uint cWarpTileXEnd = CEIL_DIV(KVtileDim, blockDim.x);
        for (uint cWarpTileYIdx = 0; cWarpTileYIdx < cWarpTileYEnd;
             ++cWarpTileYIdx) {
          //* (thread) offset Y-axis in C = Q*K^T
          const uint cTileThreadYIdx =
              cTileBlockYIdx + blockDim.y * cWarpTileYIdx + threadIdx.y;

          for (uint cWarpTileXIdx = 0; cWarpTileXIdx < cWarpTileXEnd;
               ++cWarpTileXIdx) {
            //? cTileIdxes
            //* shared memory:
            const uint cWarpTileThreadSharedMemYIdx =
                blockDim.y * cWarpTileYIdx + threadIdx.y;
            const uint cWarpTileThreadSharedMemXIdx =
                blockDim.x * cWarpTileXIdx + threadIdx.x;

            //* (thread) offset X-axis in C = Q*K^T
            const uint cTileThreadXIdx =
                cTileBlockXIdx + blockDim.x * cWarpTileXIdx + threadIdx.x;

            // scalar_t qk_acc = dscalar_zero<scalar_t>();
            float qk_acc = 0.0f;
            //! check for causality here
            // compute only the lower triangle (below main diagonal) of C =
            // Q*K^T
            if (cTileThreadXIdx <= cTileThreadYIdx) {
              for (uint i = 0; i < dimHeads; ++i) {
                qk_acc = add_g(
                    qk_acc, type2float(mul_g(
                                SMEMARRAY(qTile, dimHeads,
                                          cWarpTileThreadSharedMemYIdx, i),
                                SMEMARRAY(kTile, dimHeads,
                                          cWarpTileThreadSharedMemXIdx, i))));
#ifdef DEBUG4
                if ((blockIdx.x == 0) && (blockIdx.y == 0) &&
                    (threadIdx.x == 0) && (threadIdx.y == 3) &&
                    (cWarpTileXIdx == 0) && (kvTileIdx == 0) &&
                    (i == dimHeads - 1)) {
                  printf("qTIdx=%d|kvTIdx=%d: qTile[%d][%d] = %f\n", qTileIdx,
                         kvTileIdx, cWarpTileThreadSharedMemYIdx, i,
                         type2float(qTile[cWarpTileThreadSharedMemYIdx][i]));
                  printf("qTIdx=%d|kvTIdx=%d: kTile[%d][%d] = %f\n", qTileIdx,
                         kvTileIdx, cWarpTileThreadSharedMemXIdx, i,
                         type2float(kTile[cWarpTileThreadSharedMemXIdx][i]));
                  printf("qTIdx=%d|kvTIdx=%d: cTile[%d][%d](%d) = %f\n",
                         qTileIdx, kvTileIdx, cWarpTileThreadSharedMemYIdx,
                         cWarpTileThreadSharedMemXIdx, i, type2float(qk_acc));
                }
#endif
              }
            }
            SMEMARRAY(cTile, KVtileDim, cWarpTileThreadSharedMemYIdx,
                      cWarpTileThreadSharedMemXIdx) =
                float2type<scalar_t>(qk_acc);
            __syncthreads();
          }
        }

        //! compute H += C * V, i.e. fill hTile
        //! accumulate KVtiles to hTile
        // (QtileDim,dimHeads) = (QtileDim,KVtileDim) x (KVtileDim,dimHeads)
        // loops over hTile rows (outer) and columns (inner)
        const uint hWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
        const uint hWarpTileXEnd = CEIL_DIV(dimHeads, blockDim.x);
        for (uint hWarpTileYIdx = 0; hWarpTileYIdx < hWarpTileYEnd;
             ++hWarpTileYIdx) {
          for (uint hWarpTileXIdx = 0; hWarpTileXIdx < hWarpTileXEnd;
               ++hWarpTileXIdx) {

            //? cTileIdxes
            //* shared memory:
            const uint hWarpTileThreadSharedMemYIdx =
                blockDim.y * hWarpTileYIdx + threadIdx.y;
            const uint hWarpTileThreadSharedMemXIdx =
                blockDim.x * hWarpTileXIdx + threadIdx.x;

            // scalar_t sv_acc = dscalar_zero<scalar_t>();
            float sv_acc = 0.0f;
            for (uint i = 0; i < KVtileDim; ++i) {
              sv_acc = add_g(
                  sv_acc,
                  type2float(mul_g(SMEMARRAY(cTile, KVtileDim,
                                             hWarpTileThreadSharedMemYIdx, i),
                                   SMEMARRAY(vTile, dimHeads, i,
                                             hWarpTileThreadSharedMemXIdx))));
            }
            // accumulate over all KVtiles
            if (kvTileIdx == 0) {
              // we need to clear the hTile in first iteration
              SMEMARRAY(hTile, dimHeads, hWarpTileThreadSharedMemYIdx,
                        hWarpTileThreadSharedMemXIdx) =
                  float2type<scalar_t>(sv_acc);
            } else {
              SMEMARRAY(hTile, dimHeads, hWarpTileThreadSharedMemYIdx,
                        hWarpTileThreadSharedMemXIdx) =
                  add_g(SMEMARRAY(hTile, dimHeads, hWarpTileThreadSharedMemYIdx,
                                  hWarpTileThreadSharedMemXIdx),
                        float2type<scalar_t>(sv_acc));
            }
            __syncthreads();
          }
        }
      }

      // TODO sync all blocks here, necessary? The loop above has different
      // number of iterations for each block
      // gridGroup.sync();

      //! write hTile to global memory (has the same memory index as qTile)
      // loops over hTile rows (outer) and columns (inner)
      const uint hWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
      const uint hWarpTileXEnd = CEIL_DIV(dimHeads, blockDim.x);
      for (uint hWarpTileYIdx = 0; hWarpTileYIdx < hWarpTileYEnd;
           ++hWarpTileYIdx) {
        for (uint hWarpTileXIdx = 0; hWarpTileXIdx < hWarpTileXEnd;
             ++hWarpTileXIdx) {

          //? cTileIdxes
          //* shared memory:
          const uint hWarpTileThreadSharedMemYIdx =
              blockDim.y * hWarpTileYIdx + threadIdx.y;
          const uint hWarpTileThreadSharedMemXIdx =
              blockDim.x * hWarpTileXIdx + threadIdx.x;
          //* global memory:
          // left upper corner of cWarpTileBlock in C (global memory)
          const uint hWarpTileBlockGlobalMemIdx =
              qTileBlockGlobalMemIdx + (dimHeads * blockDim.y) * hWarpTileYIdx +
              blockDim.x * hWarpTileXIdx;
          const uint hWarpTileThreadGlobalMemIdx =
              hWarpTileBlockGlobalMemIdx + dimHeads * threadIdx.y + threadIdx.x;

          matH[hWarpTileThreadGlobalMemIdx] =
              SMEMARRAY(hTile, dimHeads, hWarpTileThreadSharedMemYIdx,
                        hWarpTileThreadSharedMemXIdx);
        }
      }
      __syncthreads();
    }
  }
}

template <typename scalar_t>
void kernel_dispatchers::vlstm_fw_dispatch(scalar_t *matH, scalar_t *matQ,
                                           scalar_t *matK, scalar_t *matV,
                                           scalar_t *iGatePreact,
                                           scalar_t *fGatePreact, int batchSize,
                                           int numHeads, int seqLen,
                                           int dimHeads) {
  printf("B: %d, NH: %d, S: %d, DH: %d\n", batchSize, numHeads, seqLen,
         dimHeads);
  const int TblockDim = TBLOCK_DIM; // matmul blockdim
  const int QtileDim = QTILE_DIM;   // blockdim for Q along seqLen dim
  const int KVtileDim = KVTILE_DIM; // blockdim for K&V along seqLen dim

  // kernel asserts
  if ((seqLen % QtileDim != 0) || (seqLen % KVtileDim != 0)) {
    printf("seqLen must be divisible by QblockDim and KVblockDim\n");
  }

  // determine the number of blocks and threads
  const dim3 blockDims(TblockDim, TblockDim);

  // TODO: determine gridDims
  // Note @mbeck: should be dynamically allocated.
  // At first parallelize across batchSize and numHeads.
  // If more streaming multiprocessors available, parallelize across seqLen.
  //! NOTE: for now we only parallelize across batchSize and numHeads
  // TODO Need to dynamically check how many blocks we can launch
  // TODO add check if batchSize*numHeads exceeds max gridDim.x

  const dim3 gridDims(batchSize * numHeads, 2);
  // const dim3 gridDims(1, 1);

  //! calculate dynamic shared memory size
  // TODO understand how memory padding works!
  // Why at innermost dim? Because memory is organized consecutively
  // we are storing the following tiles in shared memory:
  // - Input tiles: qTile, vTile, kTile -> (QtileDim, dimHeads +
  // SHARED_MEM_PADDING)
  // TODO from here add input & forgetgate tiles
  // - Intermediate result tile: cTile, dTile -> (QtileDim, KVtileDim +
  // SHARED_MEM_PADDING)
  // - Output tile: hTile -> (QtileDim, dimHeads + SHARED_MEM_PADDING)

  const uint qkvhTileSharedMemSize =
      sizeof(scalar_t) * QtileDim * (dimHeads + SHARED_MEM_PADDING);
  const uint cdTileSharedMemSize =
      sizeof(scalar_t) * QtileDim * (KVtileDim + SHARED_MEM_PADDING);

  // See here:
  // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#device-memory-accesses
  // the idea of the padding is that every number is stored in a different
  // memory bank this should help to avoid bank conflicts as many threads need
  // to access the same input and forget gate values at the same time for the
  // gate matrix computation
  // TODO check if this is really helping!
  const uint iChunkSharedMemSize =
      sizeof(scalar_t) * KVtileDim * (1 + SHARED_MEM_PADDING);
  const uint fChunkSharedMemSize =
      sizeof(scalar_t) * QtileDim * (1 + SHARED_MEM_PADDING);

  // we keep these as float as it acts as accumulator
  const uint fTileColSharedMemSize =
      sizeof(float) * QtileDim * (1 + SHARED_MEM_PADDING);
  const uint fTileColLastSharedMemSize =
      sizeof(float) * 1 * (1 + SHARED_MEM_PADDING);

  // Input/Output tiles: 4x for qTile, vTile, kTile, hTile
  // Intermediate tiles: 2x for cTile, dTile
  const uint sharedMemorySize = 4 * qkvhTileSharedMemSize +
                                2 * cdTileSharedMemSize + iChunkSharedMemSize +
                                fChunkSharedMemSize + fTileColSharedMemSize +
                                fTileColLastSharedMemSize;

  printf("blocksxy: %d-%d, threadsxy: %d-%d, shared_mem in bytes: %d\n",
         gridDims.x, gridDims.y, blockDims.x, blockDims.y, sharedMemorySize);
  // hipSetDevice(0);

  hipStream_t stream;
  hipStreamCreate(&stream);

  auto kernel = kernels::vlstm_fw<scalar_t, TblockDim, QtileDim, KVtileDim>;
  hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributePreferredSharedMemoryCarveout,
                       cudaSharedmemCarveoutMaxShared);
  hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize,
                       sharedMemorySize);
  // define void* pointers to the kernel arguments
  void *kernelArgs[] = {(void *)&matH,        (void *)&matQ,
                        (void *)&matK,        (void *)&matV,
                        (void *)&iGatePreact, (void *)&fGatePreact,
                        (void *)&batchSize,   (void *)&numHeads,
                        (void *)&seqLen,      (void *)&dimHeads};

  hipLaunchCooperativeKernel((void *)kernel, gridDims, blockDims, kernelArgs,
                              sharedMemorySize, stream);

  gpuErrchk(hipPeekAtLastError());

  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);
  gpuErrchk(hipDeviceSynchronize());

  // gpuErrchk(hipPeekAtLastError());
  // gpuErrchk(hipDeviceSynchronize());
}

// this is needed to make sure that the compiler instantiates the template
template void kernel_dispatchers::vlstm_fw_dispatch<__hip_bfloat16>(
    __hip_bfloat16 *matH, __hip_bfloat16 *matQ, __hip_bfloat16 *matK,
    __hip_bfloat16 *matV, __hip_bfloat16 *iGatePreact, __hip_bfloat16 *fGatePreact,
    int batchSize, int numHeads, int seqLen, int dimHeads);
template void kernel_dispatchers::vlstm_fw_dispatch<__half>(
    __half *matH, __half *matQ, __half *matK, __half *matV, __half *iGatePreact,
    __half *fGatePreact, int batchSize, int numHeads, int seqLen, int dimHeads);
template void kernel_dispatchers::vlstm_fw_dispatch<float>(
    float *matH, float *matQ, float *matK, float *matV, float *iGatePreact,
    float *fGatePreact, int batchSize, int numHeads, int seqLen, int dimHeads);

} // namespace vlstm