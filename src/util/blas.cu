#include "hip/hip_runtime.h"
// Copyright 2023 JKU Linz, All Rights Reserved
// Author: Korbinian Pöppel
// Adapted from the haste library
//
// See:
// Copyright 2020 LMNT, Inc. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ==============================================================================

#include "blas.h"
#include "inline_print.cuh"

hipblasStatus_t cublasHgemv2(hipblasHandle_t handle, hipblasOperation_t trans,
                            int m, int n, const __half *alpha, const __half *A,
                            int lda, const __half *x, int incx,
                            const __half *beta, __half *y, int incy) {
  float alpha_f = __half2float(*alpha);
  float beta_f = __half2float(*beta);
  return hipblasGemmEx(handle, trans, HIPBLAS_OP_N, m, 1, n, &alpha_f, A,
                      HIP_R_16F, m, x, HIP_R_16F, n, &beta_f, y, HIP_R_16F,
                      m, HIPBLAS_COMPUTE_32F_FAST_16F,
                      CUBLAS_GEMM_DFALT_TENSOR_OP);
}

hipblasStatus_t cublasHgemv3(hipblasHandle_t handle, hipblasOperation_t trans,
                            int m, int n, const __half *alpha, const __half *A,
                            int lda, const __half *x, int incx,
                            const __half *beta, __half *y, int incy) {
  return hipblasGemmEx(handle, trans, HIPBLAS_OP_N, m, 1, n, &alpha, A,
                      HIP_R_16F, m, x, HIP_R_16F, n, &beta, y, HIP_R_16F, m,
                      HIPBLAS_COMPUTE_16F, CUBLAS_GEMM_DFALT_TENSOR_OP);
}

hipblasStatus_t cublasHgemv(hipblasHandle_t handle, hipblasOperation_t trans,
                           int m, int n, const __half *alpha, const __half *A,
                           int lda, const __half *x, int incx,
                           const __half *beta, __half *y, int incy) {
  return hipblasHgemm(handle, trans, HIPBLAS_OP_N, m, 1, n, alpha, A, lda, x, n,
                     beta, y, m);
}

hipblasStatus_t cublasHgemm2(hipblasHandle_t handle, hipblasOperation_t transa,
                            hipblasOperation_t transb, int m, int n, int k,
                            const __half *alpha, /* host or device pointer */
                            const __half *A, int lda, const __half *B, int ldb,
                            const __half *beta, /* host or device pointer */
                            __half *C, int ldc) {
  float alpha_f = __half2float(*alpha);
  float beta_f = __half2float(*beta);
  return hipblasGemmEx(handle, transa, transb, m, n, k, &alpha_f, A, HIP_R_16F,
                      lda, B, HIP_R_16F, ldb, &beta_f, C, HIP_R_16F, ldc,
                      HIPBLAS_COMPUTE_32F_FAST_16F, // Compute type
                      CUBLAS_GEMM_DFALT_TENSOR_OP  // Use Tensor Cores
  );
}

hipblasStatus_t cublasHgemm3(hipblasHandle_t handle, hipblasOperation_t transa,
                            hipblasOperation_t transb, int m, int n, int k,
                            const __half *alpha, /* host or device pointer */
                            const __half *A, int lda, const __half *B, int ldb,
                            const __half *beta, /* host or device pointer */
                            __half *C, int ldc) {
  return hipblasGemmEx(handle, transa, transb, m, n, k, &alpha, A, HIP_R_16F,
                      lda, B, HIP_R_16F, ldb, &beta, C, HIP_R_16F, ldc,
                      HIPBLAS_COMPUTE_16F,         // Compute type
                      CUBLAS_GEMM_DFALT_TENSOR_OP // Use Tensor Cores
  );
}

template <typename T> __global__ void initKernel(T *data, int size, T value) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < size) {
    data[idx] = value;
  }
}

void initVector_d(hipStream_t stream, double *data, int size, double value) {
  int blockSize = 256;
  int numBlocks = (size + blockSize - 1) / blockSize;
  initKernel<<<numBlocks, blockSize, 0, stream>>>(data, size, value);
}

void initVector_f(hipStream_t stream, float *data, int size, float value) {
  int blockSize = 256;
  int numBlocks = (size + blockSize - 1) / blockSize;
  initKernel<<<numBlocks, blockSize, 0, stream>>>(data, size, value);
}

void initVector_h(hipStream_t stream, __half *data, int size, __half value) {
  int blockSize = 256;
  int numBlocks = (size + blockSize - 1) / blockSize;
  initKernel<<<numBlocks, blockSize, 0, stream>>>(data, size, value);
}

#if CUDART_VERSION >= 11020
#include <hip/hip_bf16.h>

void initVector_b(hipStream_t stream, __hip_bfloat16 *data, int size,
                  __hip_bfloat16 value) {
  int blockSize = 256;
  int numBlocks = (size + blockSize - 1) / blockSize;
  initKernel<<<numBlocks, blockSize, 0, stream>>>(data, size, value);
}

hipblasStatus_t cublasBgemv(hipblasHandle_t handle, hipblasOperation_t trans,
                           int m, int n, const __hip_bfloat16 *alpha,
                           const __hip_bfloat16 *A, int lda,
                           const __hip_bfloat16 *x, int incx,
                           const __hip_bfloat16 *beta, __hip_bfloat16 *y,
                           int incy) {
  float alpha_f = __bfloat162float(*alpha);
  float beta_f = __bfloat162float(*beta);
  return hipblasGemmEx(handle, trans, HIPBLAS_OP_N, m, 1, n, &alpha_f, A,
                      HIP_R_16BF, m, x, HIP_R_16BF, n, &beta_f, y,
                      HIP_R_16BF, m, HIPBLAS_COMPUTE_32F_FAST_16F,
                      CUBLAS_GEMM_DFALT_TENSOR_OP);
}

hipblasStatus_t
cublasBgemm(hipblasHandle_t handle, hipblasOperation_t transa,
            hipblasOperation_t transb, int m, int n, int k,
            const __hip_bfloat16 *alpha, /* host or device pointer */
            const __hip_bfloat16 *A, int lda, const __hip_bfloat16 *B, int ldb,
            const __hip_bfloat16 *beta, /* host or device pointer */
            __hip_bfloat16 *C, int ldc) {
  float alpha_f = __bfloat162float(*alpha);
  float beta_f = __bfloat162float(*beta);
  return hipblasGemmEx(handle, transa, transb, m, n, k, &alpha_f, A, HIP_R_16BF,
                      lda, B, HIP_R_16BF, ldb, &beta_f, C, HIP_R_16BF, ldc,
                      HIPBLAS_COMPUTE_32F_FAST_16F, // Compute type
                      CUBLAS_GEMM_DFALT_TENSOR_OP  // Use Tensor Cores
  );
}

#endif
