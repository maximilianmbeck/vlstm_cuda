#include "hip/hip_runtime.h"
// Copyright JKU Linz 2024
// Author: Maximilian Beck

#include <hip/hip_cooperative_groups.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <hip/hip_math_constants.h>
#include <sys/types.h>

#include "../util/cuda_errorcheck.h"
#include "../util/inline_ops.cuh"
#include "../util/inline_print.cuh"
#include "../util/support.h"
#include "kernel_dispatchers.h"

#define CEIL_DIV(a, b) (((a) + (b)-1) / (b))
#define FLOOR_DIV(a, b) ((a) / (b))
#define MAX(a, b) (((a) > (b)) ? (a) : (b))
#define MIN(a, b) (((a) < (b)) ? (a) : (b))

namespace vlstm {

namespace cg = cooperative_groups;

namespace kernels {

template <typename scalar_t, int TblockDim, int QblockDim, int KVblockDim>
__global__ void vlstm_bw(scalar_t *deltaQ, scalar_t *deltaK, scalar_t *deltaV,
                         scalar_t *deltaIGatePreact, scalar_t *deltaFGatePreact,
                         scalar_t *matC, scalar_t *deltaH, scalar_t *matQ,
                         scalar_t *matK, scalar_t *matV, scalar_t *iGatePreact,
                         scalar_t *fGatePreact, scalar_t *vecN, scalar_t *vecM,
                         float *csDeltaDTildeChunkArr, float *csDeltaDTildeVec,
                         int batchSize, int numHeads, int seqLen, int dimHeads);

} // namespace kernels

////////////////////////////////////////////////////////////////////////////////////////

#define TBLOCK_DIM 4 // TblockDim: corresponds to BLOCK_DIM in matmul
#define KVTILE_DIM 8 // KVtileDim: TileDim for K&V along seqLen dim
// QTILE_DIM must be divisible by KVTILE_DIM and TBLOCK_DIM,
// KVTILE_DIM <= QTILE_DIM
#define QTILE_DIM 8 // QtileDim: TileDim for Q along seqLen dim

// shared memory must be aligned: depends on scalar_t (multiples of 4 should be
// fine for bf16, fp16 and fp32)
#define SHARED_MEM_PADDING 8 // SHARED_MEM_PADDING: padding for shared memory

// SMEMARRAY: access shared memory array (2D)
#define SMEMARRAY(array, stride, row, col)                                     \
  array[(row) * (stride + SHARED_MEM_PADDING) + (col)]
// SMEMVECTOR: access shared memory vector (1D)
#define SMEMVECTOR(array, idx) array[(idx) * (1 + SHARED_MEM_PADDING)]

#define DEBUG 1
// #define OUTPUTdDTile 1
// #define OUTPUTdDtildeTile 1
// #define OUTPUTDTile 1
// #define OUTPUTDcsTile 1
#define OUTPUTPRTile 1
#define OUTPUTPRTileR 1

// #define DEBUG_WRdeltaI 1
// #define DEBUG_deltaISUM0 1
// #define DEBUG_deltaISUM1 1
// #define DEBUG_deltaISUM2 1

// #define DEBUG_IJIDX 1

// #define DEBUG_DeltaDCS0 1
// #define DEBUG_DeltaDCS1 1
// #define DEBUG_DeltaDCS2 1

/**
Conventions:
- chunk: A 1D vector in shared memory
- tile: A 2D matrix in shared memory
*/

/* vLSTM Backward Kernel v0 */

template <typename scalar_t, int TblockDim, int QtileDim, int KVtileDim>
__global__ void
kernels::vlstm_bw(scalar_t *deltaQ, scalar_t *deltaK, scalar_t *deltaV,
                  scalar_t *deltaIGatePreact, scalar_t *deltaFGatePreact,
                  scalar_t *matC, scalar_t *deltaH, scalar_t *matQ,
                  scalar_t *matK, scalar_t *matV, scalar_t *iGatePreact,
                  scalar_t *fGatePreact, scalar_t *vecN, scalar_t *vecM,
                  float *csDeltaDTildeChunkArr, float *csDeltaDTildeVec,
                  int batchSize, int numHeads, int seqLen, int dimHeads) {
  // int tIdx = threadIdx.x + blockDim.x * threadIdx.y;
#ifdef DEBUG
  if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
      (threadIdx.y == 0)) {
    printf(
        "In BW-Kernel: gdim.x: %d, gdim.y: %d, gdim.z: %d, bdim.x: %d, bdim.y: "
        "%d\n",
        gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y);
  }
#endif

#ifdef DEBUG
  if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
      (threadIdx.y == 0)) {
    printf("In BW-Kernel: QtileDim: %d, KVtileDim: %d, TblockDim:%d\n",
           QtileDim, KVtileDim, TblockDim);
  }
#endif
  // TODO can we define multiple gridGroups?
  // -> e.g. all threadblocks with the same gridDim.x should go into one group
  cg::grid_group gridGroup = cg::this_grid();

  //! Shared Memory aka SRAM
  // the data in this shared memory is shared across all threads in a thread
  // block
  extern __shared__ float sbuf[]; // declare it as float and redefine it later
  // Note: keep in mind the memory is defined in a contiguous region.
  // One pointer has the full memory space until the next point is defined.
  // Therefore to read the size of a single shared memory array you need to
  // have a look at the offset for the next array.

  //? input-output tiles
  //* (QtileDim x dimHeads) tiles:
  // qTile (QTileDim x dimHeads)
  scalar_t *qTile = (scalar_t *)sbuf;
  // deltaQTile (QTileDim x dimHeads)
  scalar_t *deltaQTile =
      (scalar_t *)&qTile[QtileDim * (dimHeads + SHARED_MEM_PADDING)];
  // deltaHTile (QTileDim x dimHeads)
  scalar_t *deltaHTile =
      (scalar_t *)&deltaQTile[QtileDim * (dimHeads + SHARED_MEM_PADDING)];

  //* (QtileDim x 1) chunks:
  // nChunk (QTileDim x 1)
  scalar_t *nChunk =
      (scalar_t *)&deltaHTile[QtileDim * (dimHeads + SHARED_MEM_PADDING)];
  // mDeltaDcsCorrChunk (QTileDim x 1)
  scalar_t *mDeltaDcsCorrChunk =
      (scalar_t *)&nChunk[QtileDim * (1 + SHARED_MEM_PADDING)];

  //* (KVtileDim x dimHeads) tiles:
  // kTile (KVtileDim x dimHeads)
  scalar_t *kTile =
      (scalar_t *)&mDeltaDcsCorrChunk[QtileDim * (1 + SHARED_MEM_PADDING)];
  // vTile (KVtileDim x dimHeads)
  scalar_t *vTile =
      (scalar_t *)&kTile[KVtileDim * (dimHeads + SHARED_MEM_PADDING)];
  // deltaKTile (KVtileDim x dimHeads)
  scalar_t *deltaKTile =
      (scalar_t *)&vTile[KVtileDim * (dimHeads + SHARED_MEM_PADDING)];
  // deltaVTile (KVtileDim x dimHeads)
  scalar_t *deltaVTile =
      (scalar_t *)&deltaKTile[KVtileDim * (dimHeads + SHARED_MEM_PADDING)];

  //* (KVtileDim x 1) chunks:
  // iChunk (KVtileDim x 1)
  scalar_t *iChunk =
      (scalar_t *)&deltaVTile[KVtileDim * (dimHeads + SHARED_MEM_PADDING)];
  // fChunk (KVtileDim x 1)
  scalar_t *fChunk = (scalar_t *)&iChunk[KVtileDim * (1 + SHARED_MEM_PADDING)];
  // deltaIChunk (KVtileDim x 1)
  scalar_t *deltaIChunk =
      (scalar_t *)&fChunk[QtileDim * (1 + SHARED_MEM_PADDING)];
  // deltaFChunk (KVtileDim x 1)
  scalar_t *deltaFChunk =
      (scalar_t *)&deltaIChunk[KVtileDim * (1 + SHARED_MEM_PADDING)];

  //? intermediate tiles
  //* (QtileDim x KVtileDim) tiles:
  // dstrRTile (QtileDim x KVtileDim)
  scalar_t *dstrRTile =
      (scalar_t *)&deltaFChunk[KVtileDim * (1 + SHARED_MEM_PADDING)];
  // sTile (QtileDim x KVtileDim)
  scalar_t *sPTile =
      (scalar_t *)&dstrRTile[KVtileDim * (1 + SHARED_MEM_PADDING)];
  // dDTile (QtileDim x KVtileDim)
  scalar_t *dDTile =
      (scalar_t *)&sPTile[QtileDim * (KVtileDim + SHARED_MEM_PADDING)];
  // dCDcsTile (QtileDim x KVtileDim)
  scalar_t *dCDcsTile =
      (scalar_t *)&dDTile[QtileDim * (KVtileDim + SHARED_MEM_PADDING)];

  //* (KVtileDim x 1) chunks:
  float *fAccRowChunk =
      (float *)(&(dCDcsTile[QtileDim * (KVtileDim + SHARED_MEM_PADDING)]));

  //? flatten the threads to 1D
  const uint flatThreadIdx = blockDim.x * threadIdx.y + threadIdx.x;

  //! PARALLELIZE ALONG BATCHSIZE * NUMHEADS (gridDim.x)
  const uint batchHeadStepQKVdH = seqLen * dimHeads;
  const uint batchHeadStepIFNMgate =
      seqLen * 1; // TODO rename: csDeltaDTildeVec has same step
  const uint batchHeadStepCD = seqLen * seqLen;
  const uint batchHeadStepDeltaDcsChunkArr = gridDim.y * QtileDim;
  const uint numBatchHeads = batchSize * numHeads;
  // End for looplevel 0:
  const uint batchHeadEnd = CEIL_DIV(numBatchHeads, gridDim.x);
  //! looplevel 0: loop over batches and heads
  for (uint batchHeadIdx = 0; batchHeadIdx < batchHeadEnd; ++batchHeadIdx) {

    // dQ, dK, dV also have this index
    const uint batchHeadGridXGlobalMemIdxQKVdH =
        (batchHeadStepQKVdH * gridDim.x) * batchHeadIdx +
        (batchHeadStepQKVdH)*blockIdx.x;

    const uint batchHeadGridXGlobalMemIdxIFNMgate =
        (batchHeadStepIFNMgate * gridDim.x) * batchHeadIdx +
        (batchHeadStepIFNMgate)*blockIdx.x;

    const uint batchHeadGridXGlobalMemIdxCD =
        (batchHeadStepCD * gridDim.x) * batchHeadIdx +
        (batchHeadStepCD)*blockIdx.x;

    const uint batchHeadGridXGlobalMemIdxDeltaDcsChunkArr =
        (batchHeadStepDeltaDcsChunkArr * gridDim.x) * batchHeadIdx +
        (batchHeadStepDeltaDcsChunkArr)*blockIdx.x;

#ifdef DEBUG5
    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
      printf("B<%d,%d> batchHeadIdx: %d, batchHeadEnd: %d, "
             "batchHeadGridXGlobalMemIdxQKV: "
             "%d\n",
             blockIdx.x, blockIdx.y, batchHeadIdx, batchHeadEnd,
             batchHeadGridXGlobalMemIdxQKV);
    }
#endif

    //! Initialize deltaQ, deltaK, deltaV to zero in HBM
    // already done in the kernel dispatcher, via torch::zeros()

    //! Initialize csDeltaDTildeVec to zero in HBM
    // already done in the kernel dispatcher, via hipMemset

    //! PARALLELIZE ALONG SEQLEN (gridDim.y)
    //! looplevel 1 (j-loop): loop over KVtile blocks along seqLen dim
    // Ends for looplevel 1:
    const uint kvTileEnd = CEIL_DIV(seqLen, KVtileDim * gridDim.y);
    for (uint kvTileIdx = 0; kvTileIdx < kvTileEnd; ++kvTileIdx) {

      //* kTile, vTile Global Memory Index
      // (grid&block) offset in K,V matrix for kTile&vTile (global memory)
      const uint kvdKdVTileGridXYGlobalMemIdx =
          batchHeadGridXGlobalMemIdxQKVdH +
          (dimHeads * KVtileDim * gridDim.y) * kvTileIdx;
      const uint kvdKdVTileBlockGlobalMemIdx =
          kvdKdVTileGridXYGlobalMemIdx + (dimHeads * KVtileDim) * blockIdx.y;

      //* sTile Global Memory Index (virtual, as never materialized fully)
      // (grid&block) offset Y-axis in S = Q*K^T matrix (along sequence
      // dimension) (used for checking causality)
      // Note: we add "Xdim" to the name to indicate that this is the
      // corresponding x/column-dimension in the S matrix (differs from the
      // gridDim.y)
      const uint sTileXdimGridYIdx = KVtileDim * gridDim.y * kvTileIdx;
      const uint sTileXdimBlockYIdx =
          sTileXdimGridYIdx + KVtileDim * blockIdx.y;

      const uint iChunkGridXYGlobalMemIdx =
          batchHeadGridXGlobalMemIdxIFNMgate +
          (1 * KVtileDim * gridDim.y) * kvTileIdx;
      const uint iChunkBlockGlobalMemIdx =
          iChunkGridXYGlobalMemIdx + (1 * KVtileDim) * blockIdx.y;

      //! Load iChunk, Init deltaIChunk, deltaFChunk & fAccRowChunk to
      //! zero in SRAM
      const uint idFdIChunkEnd = CEIL_DIV(KVtileDim, blockDim.x * blockDim.y);
      for (uint idFdIChunkIdx = 0; idFdIChunkIdx < idFdIChunkEnd;
           ++idFdIChunkIdx) {
        //? idFdI idxes
        //* shared memory
        const uint idFdIThreadSharedMemIdx =
            flatThreadIdx + blockDim.x * blockDim.y * idFdIChunkIdx;
        //* global memory
        const uint iThreadGlobalMemIdx =
            iChunkBlockGlobalMemIdx + flatThreadIdx;

        if (idFdIThreadSharedMemIdx < KVtileDim) {
          SMEMVECTOR(iChunk, idFdIThreadSharedMemIdx) =
              iGatePreact[iThreadGlobalMemIdx];
          SMEMVECTOR(deltaIChunk, idFdIThreadSharedMemIdx) =
              dscalar_zero<scalar_t>();
          SMEMVECTOR(deltaFChunk, idFdIThreadSharedMemIdx) =
              dscalar_zero<scalar_t>();
          SMEMVECTOR(fAccRowChunk, idFdIThreadSharedMemIdx) = 0.0f;
        }
      }
      __syncthreads();

      //! Load kTile & vTile, Init deltaKTile & deltaVTile to zero in SRAM
      // loops over rows (outer) and columns (inner) of kTile and vTile
      const uint kvWarpTileYEnd = CEIL_DIV(KVtileDim, blockDim.y); // rows
      const uint kvWarpTileXEnd = CEIL_DIV(dimHeads, blockDim.x);  // cols
      for (uint kvWarpTileYIdx = 0; kvWarpTileYIdx < kvWarpTileYEnd;
           ++kvWarpTileYIdx) {
#ifdef DEBUG2
        if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0) &&
            (threadIdx.y == 0)) {
          printf("kvWarpTileYIdx=%d: kvWarpTileYEnd: %d, kvWarpTileXEnd: %d\n",
                 kvWarpTileYIdx, kvWarpTileYEnd, kvWarpTileXEnd);
        }
#endif
        for (uint kvWarpTileXIdx = 0; kvWarpTileXIdx < kvWarpTileXEnd;
             ++kvWarpTileXIdx) {
          //? kvWarpTileIdxes for k-tile AND v-tile
          //* shared memory:
          const uint kvWarpTileThreadSharedMemYIdx =
              blockDim.y * kvWarpTileYIdx + threadIdx.y;
          const uint kvWarpTileThreadSharedMemXIdx =
              blockDim.x * kvWarpTileXIdx + threadIdx.x;
          //* global memory:
          // left upper corner of kTileBlock in K (global memory)
          const uint kvWarpTileBlockGlobalMemIdx =
              kvdKdVTileBlockGlobalMemIdx +
              (dimHeads * blockDim.y) * kvWarpTileYIdx +
              blockDim.x * kvWarpTileXIdx;
          const uint kvWarpTileThreadGlobalMemIdx =
              kvWarpTileBlockGlobalMemIdx + dimHeads * threadIdx.y +
              threadIdx.x;

          //! For simplicity: assume that KVtileDim is a multiple of blockDim.y
          //! and dimHeads is a multiple of blockDim.x
          SMEMARRAY(kTile, dimHeads, kvWarpTileThreadSharedMemYIdx,
                    kvWarpTileThreadSharedMemXIdx) =
              matK[kvWarpTileThreadGlobalMemIdx];
          SMEMARRAY(vTile, dimHeads, kvWarpTileThreadSharedMemYIdx,
                    kvWarpTileThreadSharedMemXIdx) =
              matV[kvWarpTileThreadGlobalMemIdx];

          // init deltaKTile and deltaVTile to zero
          SMEMARRAY(deltaKTile, dimHeads, kvWarpTileThreadSharedMemYIdx,
                    kvWarpTileThreadSharedMemXIdx) = dscalar_zero<scalar_t>();
          SMEMARRAY(deltaVTile, dimHeads, kvWarpTileThreadSharedMemYIdx,
                    kvWarpTileThreadSharedMemXIdx) = dscalar_zero<scalar_t>();
        }
      }
      __syncthreads();

      //! looplevel 2 (i-loop): loop over QTile blocks along seqLen dim
      const uint qTileEnd = CEIL_DIV(seqLen, QtileDim);
      const uint jIdx = blockIdx.y + kvTileIdx * gridDim.y;
      //   const uint qTileStart = FLOOR_DIV(jIdx * KVtileDim, QtileDim);
      //? We start from the first QTile such that we can sync the threadblocks
      //? globally, and ensure that they are in sync for the next KVtile
      //? they need to be in sync for cumsum(deltaDtildeTile) and qTile
      const uint qTileStart = 0;
      for (uint qTileIdx = qTileStart; qTileIdx < qTileEnd; ++qTileIdx) {

        const uint iIdx = qTileIdx;
        //* qTile Global Memory Index
        const uint qdHdQTileBlockGlobalMemIdx =
            batchHeadGridXGlobalMemIdxQKVdH + (dimHeads * QtileDim) * qTileIdx;

        //* nChunk, mDeltaDcsCorrChunk, fChunk Global Memory Index
        const uint nmfChunkBlockGlobalMemIdx =
            batchHeadGridXGlobalMemIdxIFNMgate + (1 * QtileDim) * qTileIdx;

        //* sTile Global Memory Index
        const uint sTileYdimGridYIdx = QtileDim * qTileIdx;
        const uint sTileYdimBlockYIdx = sTileYdimGridYIdx;

#ifdef DEBUG_IJIDX
        if ((blockIdx.x == 0) && (blockIdx.y <= 1) && (flatThreadIdx == 0)) {
          printf("blockIdx(x,y)=(%d,%d), ijIdx(i,j)=(%d,%d), "
                 "sTileXYIdx(x,y)=(%d,%d), kvTileIdx=%d, qTileIdx=%d\n",
                 blockIdx.x, blockIdx.y, iIdx, jIdx, sTileXdimBlockYIdx,
                 sTileYdimBlockYIdx, kvTileIdx, qTileIdx);
        }
#endif

        //! Load nChunk, mDeltaDcsCorrChunk, fChunk in SRAM
        const uint nmfChunkEnd = CEIL_DIV(QtileDim, blockDim.x * blockDim.y);
        for (uint nmfChunkIdx = 0; nmfChunkIdx < nmfChunkEnd; ++nmfChunkIdx) {
          //? nmf idxes
          //* shared memory
          const uint nmfThreadSharedMemIdx =
              flatThreadIdx + blockDim.x * blockDim.y * nmfChunkIdx;
          //* global memory
          const uint nmfThreadGlobalMemIdx =
              nmfChunkBlockGlobalMemIdx + flatThreadIdx;

          if (nmfThreadSharedMemIdx < QtileDim) {
            SMEMVECTOR(nChunk, nmfThreadSharedMemIdx) =
                vecN[nmfThreadGlobalMemIdx];
            SMEMVECTOR(mDeltaDcsCorrChunk, nmfThreadSharedMemIdx) =
                vecM[nmfThreadGlobalMemIdx];

            // load raw preactivations for D matrix and deltaF
            SMEMVECTOR(fChunk, nmfThreadSharedMemIdx) =
                fGatePreact[nmfThreadGlobalMemIdx];
          }
        }

        //! Load qTile & deltaHTile in SRAM
        // (QTileDim x dimHeads)
        // loops over rows (outer) and columns (inner) of qTile and deltaHTile
        const uint qdHWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y); // rows
        const uint qdHWarpTileXEnd = CEIL_DIV(dimHeads, blockDim.x); // cols
        for (uint qdHWarpTileYIdx = 0; qdHWarpTileYIdx < qdHWarpTileYEnd;
             ++qdHWarpTileYIdx) {
          for (uint qdHWarpTileXIdx = 0; qdHWarpTileXIdx < qdHWarpTileXEnd;
               ++qdHWarpTileXIdx) {
            //? qdHWarpTileIdxes for q-tile AND delta-h-tile
            //* shared memory:
            const uint qdHWarpTileThreadSharedMemYIdx =
                blockDim.y * qdHWarpTileYIdx + threadIdx.y;
            const uint qdHWarpTileThreadSharedMemXIdx =
                blockDim.x * qdHWarpTileXIdx + threadIdx.x;
            //* global memory:
            // left upper corner of qTileBlock in Q (global memory)
            const uint qdHWarpTileBlockGlobalMemIdx =
                qdHdQTileBlockGlobalMemIdx +
                (dimHeads * blockDim.y) * qdHWarpTileYIdx +
                blockDim.x * qdHWarpTileXIdx;
            const uint qdHWarpTileThreadGlobalMemIdx =
                qdHWarpTileBlockGlobalMemIdx + dimHeads * threadIdx.y +
                threadIdx.x;
            SMEMARRAY(qTile, dimHeads, qdHWarpTileThreadSharedMemYIdx,
                      qdHWarpTileThreadSharedMemXIdx) =
                matQ[qdHWarpTileThreadGlobalMemIdx];
            SMEMARRAY(deltaHTile, dimHeads, qdHWarpTileThreadSharedMemYIdx,
                      qdHWarpTileThreadSharedMemXIdx) =
                deltaH[qdHWarpTileThreadGlobalMemIdx];
          }
        }
        __syncthreads();
        //? Global Sync
        gridGroup.sync();

        //! Compute deltaCTile = (deltaHtile  vTile^T) / nChunk (and divide by
        //! nChunk)
        // (QtileDim,KVtileDim) = (QtileDim,dimHeads) x (dimHeads,KVtileDim)
        // loops over cTile rows (outer) and columns (inner)
        const uint dCWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
        const uint dCWarpTileXEnd = CEIL_DIV(KVtileDim, blockDim.x);
        for (uint dCWarpTileYIdx = 0; dCWarpTileYIdx < dCWarpTileYEnd;
             ++dCWarpTileYIdx) {

          for (uint dCWarpTileXIdx = 0; dCWarpTileXIdx < dCWarpTileXEnd;
               ++dCWarpTileXIdx) {
            //? cTileIdxes
            //* shared memory:
            const uint cWarpTileThreadSharedMemYIdx =
                blockDim.y * dCWarpTileYIdx + threadIdx.y;
            const uint cWarpTileThreadSharedMemXIdx =
                blockDim.x * dCWarpTileXIdx + threadIdx.x;

            // scalar_t qk_acc = dscalar_zero<scalar_t>();
            float acc = 0.0f;
            for (uint i = 0; i < dimHeads; ++i) {
              acc =
                  add_g(acc, type2float(mul_g(
                                 SMEMARRAY(deltaHTile, dimHeads,
                                           cWarpTileThreadSharedMemYIdx, i),
                                 SMEMARRAY(vTile, dimHeads,
                                           cWarpTileThreadSharedMemXIdx, i))));
            }
            // dC = deltaH *V^T / n
            scalar_t nChunkVal =
                SMEMVECTOR(nChunk, cWarpTileThreadSharedMemYIdx);

            // we first cast to scalar_t and then divide (this will also the
            // case for tensor cores)
            SMEMARRAY(dCDcsTile, KVtileDim, cWarpTileThreadSharedMemYIdx,
                      cWarpTileThreadSharedMemXIdx) =
                div_g(float2type<scalar_t>(acc), nChunkVal);
          }
        }
        __syncthreads();

        //! Compute sTile = (qTile  kTile^T) * (1/sqrt(d)) and
        //! dDTile = deltaCTile * sTile (pointwise)
        // (QtileDim,KVtileDim) = (QtileDim,dimHeads) x (dimHeads,KVtileDim)
        // loops over sTile rows (outer) and columns (inner)
        const uint sWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
        const uint sWarpTileXEnd = CEIL_DIV(KVtileDim, blockDim.x);
        for (uint sWarpTileYIdx = 0; sWarpTileYIdx < sWarpTileYEnd;
             ++sWarpTileYIdx) {

          for (uint sWarpTileXIdx = 0; sWarpTileXIdx < sWarpTileXEnd;
               ++sWarpTileXIdx) {
            //? cTileIdxes
            //* shared memory:
            const uint sWarpTileThreadSharedMemYIdx =
                blockDim.y * sWarpTileYIdx + threadIdx.y;
            const uint sWarpTileThreadSharedMemXIdx =
                blockDim.x * sWarpTileXIdx + threadIdx.x;

            // scalar_t qk_acc = dscalar_zero<scalar_t>();
            float acc = 0.0f;
            for (uint i = 0; i < dimHeads; ++i) {
              acc =
                  add_g(acc, type2float(mul_g(
                                 SMEMARRAY(qTile, dimHeads,
                                           sWarpTileThreadSharedMemYIdx, i),
                                 SMEMARRAY(kTile, dimHeads,
                                           sWarpTileThreadSharedMemXIdx, i))));
            }
            // compute sTile
            scalar_t s_val =
                float2type<scalar_t>(mul_g(acc, rsqrtf(type2float(dimHeads))));
            SMEMARRAY(sPTile, KVtileDim, sWarpTileThreadSharedMemYIdx,
                      sWarpTileThreadSharedMemXIdx) = s_val;
            // compute dDTile
            scalar_t deltaC_val =
                SMEMARRAY(dCDcsTile, KVtileDim, sWarpTileThreadSharedMemYIdx,
                          sWarpTileThreadSharedMemXIdx);
            scalar_t ddd_val = mul_g(deltaC_val, s_val);
            SMEMARRAY(dDTile, KVtileDim, sWarpTileThreadSharedMemYIdx,
                      sWarpTileThreadSharedMemXIdx) = ddd_val;
          }
        }
        __syncthreads();

        //! Compute dDtile = deltaCTile * sTile
        // Done with the pointwise multiplication in the previous step

#ifdef OUTPUTdDTile
        //! DEBUG: write dDtile to global memory
        // left upper corner of cWarpTileBlock in C (global memory)
        //* cdTile Global Memory Index (Debug only)
        const uint cdTileGridXYGlobalMemIdx =
            batchHeadGridXGlobalMemIdxCD + (seqLen * QtileDim) * qTileIdx;
        const uint cdTileBlockGlobalMemIdx =
            cdTileGridXYGlobalMemIdx + (kvTileIdx * KVtileDim * gridDim.y) +
            (1 * KVtileDim) * blockIdx.y;

        const uint cdWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
        const uint cdWarpTileXEnd = CEIL_DIV(KVtileDim, blockDim.x);
        for (uint cdWarpTileYIdx = 0; cdWarpTileYIdx < cdWarpTileYEnd;
             ++cdWarpTileYIdx) {
          for (uint cdWarpTileXIdx = 0; cdWarpTileXIdx < cdWarpTileXEnd;
               ++cdWarpTileXIdx) {
            //? cTileIdxes
            //* shared memory:
            const uint cdWarpTileThreadSharedMemYIdx =
                blockDim.y * cdWarpTileYIdx + threadIdx.y;
            const uint cdWarpTileThreadSharedMemXIdx =
                blockDim.x * cdWarpTileXIdx + threadIdx.x;
            //* global memory:
            const uint cdWarpTileBlockGlobalMemIdx =
                cdTileBlockGlobalMemIdx +
                (seqLen * blockDim.y) * cdWarpTileYIdx +
                blockDim.x * cdWarpTileXIdx;
            const uint cdWarpTileThreadGlobalMemIdx =
                cdWarpTileBlockGlobalMemIdx + seqLen * threadIdx.y +
                threadIdx.x;

            matC[cdWarpTileThreadGlobalMemIdx] =
                SMEMARRAY(dDTile, KVtileDim, cdWarpTileThreadSharedMemYIdx,
                          cdWarpTileThreadSharedMemXIdx);
          }
        }
#endif
        //! Construct D'Tile from fChunk and iChunk and
        //! compute deltaDtildeTile = deltaDTile * D'Tile
        // flatten all threads to 1D along kvTileDim (j-direction),
        // sum up the f gate values in i-direction (qTileDim),
        // store the last row of the D'Tile in fAccRowChunk
        // take care of causality

        // loop in j-direction (kvTileDim / x-dim)
        const uint dTileXdimEnd = CEIL_DIV(KVtileDim, blockDim.x * blockDim.y);
        for (uint dTileXdimIdx = 0; dTileXdimIdx < dTileXdimEnd;
             ++dTileXdimIdx) {
          //? dTile idxes
          //* shared memory
          const uint dTileXdimThreadSharedMemIdx =
              flatThreadIdx + blockDim.x * blockDim.y * dTileXdimIdx;

          //* dTile global index (virtual, as never materialized fully)
          const uint dTileXdimThreadIdx =
              sTileXdimBlockYIdx + dTileXdimThreadSharedMemIdx;

          if (dTileXdimThreadSharedMemIdx < KVtileDim) {
            // pre load the i_val for the current column
            const scalar_t i_val =
                SMEMVECTOR(iChunk, dTileXdimThreadSharedMemIdx);

            // sum up f gate values in i-direction
            float f_acc = SMEMVECTOR(fAccRowChunk, dTileXdimThreadSharedMemIdx);
            // loop in j-direction (qTileDim / y-dim)
            for (uint dTileYdimThreadSharedMemIdx = 0;
                 dTileYdimThreadSharedMemIdx < QtileDim;
                 ++dTileYdimThreadSharedMemIdx) {

              //* dTile global index (virtual, as never materialized fully)
              const uint dTileYdimThreadIdx =
                  sTileYdimBlockYIdx + dTileYdimThreadSharedMemIdx;

              //? Compute f gate cumsum entries in dTile
              // only sum up the f gates in the lower triangular (take care of
              // causality)
              if (dTileYdimThreadIdx > dTileXdimThreadIdx) {
                scalar_t f_val = logsigmoid_g(
                    SMEMVECTOR(fChunk, dTileYdimThreadSharedMemIdx));
                f_acc = add_g(f_acc, type2float(f_val));
              }
              // store the last row of D'Tile in fAccRowChunk
              if (dTileYdimThreadSharedMemIdx == QtileDim - 1) {
                SMEMVECTOR(fAccRowChunk, dTileXdimThreadSharedMemIdx) = f_acc;
              }

              //? Create D'Tile entries sum(f) + i
              //? Create deltaDtildeTile entries (overwrite the dDTile entries)
              scalar_t d_val = dscalar_zero<scalar_t>();
              scalar_t deltaDtilde_val = dscalar_zero<scalar_t>();
              if (dTileYdimThreadIdx < dTileXdimThreadIdx) {
                // (-> in upper triangular part)
                // d_val = 0;
              } else {
                scalar_t deltaD_val =
                    SMEMARRAY(dDTile, KVtileDim, dTileYdimThreadSharedMemIdx,
                              dTileXdimThreadSharedMemIdx);
                scalar_t m_val =
                    SMEMVECTOR(mDeltaDcsCorrChunk, dTileYdimThreadSharedMemIdx);

                if (dTileYdimThreadIdx == dTileXdimThreadIdx) {
                  d_val = exp_g(sub_g(i_val, m_val));
                } else {
                  // (dTileYdimThreadIdx > dTileXdimThreadIdx)
                  // (-> in lower triangular part)
                  scalar_t dtilde_val =
                      float2type<scalar_t>(add_g(f_acc, type2float(i_val)));
                  d_val = exp_g(sub_g(dtilde_val, m_val));
                }
                deltaDtilde_val = mul_g(deltaD_val, d_val);
              }
              // store the D'Tile entries in dstrRTile
              SMEMARRAY(dstrRTile, KVtileDim, dTileYdimThreadSharedMemIdx,
                        dTileXdimThreadSharedMemIdx) = d_val;
              // store the deltaDtildeTile entries in dDTile
              SMEMARRAY(dDTile, KVtileDim, dTileYdimThreadSharedMemIdx,
                        dTileXdimThreadSharedMemIdx) = deltaDtilde_val;

            } // end for (dTileYdimThreadSharedMemIdx)
          }   // end if (dTileXdimThreadSharedMemIdx < KVtileDim)
        }     // end for (dTileXdimIdx)
        __syncthreads();

#ifdef OUTPUTDTile
        //! DEBUG: write D'tile to global memory
        // left upper corner of cWarpTileBlock in C (global memory)
        //* cdTile Global Memory Index (Debug only)
        const uint cdTileGridXYGlobalMemIdx =
            batchHeadGridXGlobalMemIdxCD + (seqLen * QtileDim) * qTileIdx;
        const uint cdTileBlockGlobalMemIdx =
            cdTileGridXYGlobalMemIdx + (kvTileIdx * KVtileDim * gridDim.y) +
            (1 * KVtileDim) * blockIdx.y;

        const uint cdWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
        const uint cdWarpTileXEnd = CEIL_DIV(KVtileDim, blockDim.x);
        for (uint cdWarpTileYIdx = 0; cdWarpTileYIdx < cdWarpTileYEnd;
             ++cdWarpTileYIdx) {
          for (uint cdWarpTileXIdx = 0; cdWarpTileXIdx < cdWarpTileXEnd;
               ++cdWarpTileXIdx) {
            //? cTileIdxes
            //* shared memory:
            const uint cdWarpTileThreadSharedMemYIdx =
                blockDim.y * cdWarpTileYIdx + threadIdx.y;
            const uint cdWarpTileThreadSharedMemXIdx =
                blockDim.x * cdWarpTileXIdx + threadIdx.x;
            //* global memory:
            const uint cdWarpTileBlockGlobalMemIdx =
                cdTileBlockGlobalMemIdx +
                (seqLen * blockDim.y) * cdWarpTileYIdx +
                blockDim.x * cdWarpTileXIdx;
            const uint cdWarpTileThreadGlobalMemIdx =
                cdWarpTileBlockGlobalMemIdx + seqLen * threadIdx.y +
                threadIdx.x;

            matC[cdWarpTileThreadGlobalMemIdx] =
                SMEMARRAY(dstrRTile, KVtileDim, cdWarpTileThreadSharedMemYIdx,
                          cdWarpTileThreadSharedMemXIdx);
          }
        }
#endif
        //! Compute deltaDtildeTile = deltaDTile * D'Tile
        // Computed with pointwise multiplication in the previous step

#ifdef OUTPUTdDtildeTile
        //! DEBUG: write Dtilde Tile to global memory
        // left upper corner of cWarpTileBlock in C (global memory)
        //* cdTile Global Memory Index (Debug only)
        const uint cdTileGridXYGlobalMemIdx =
            batchHeadGridXGlobalMemIdxCD + (seqLen * QtileDim) * qTileIdx;
        const uint cdTileBlockGlobalMemIdx =
            cdTileGridXYGlobalMemIdx + (kvTileIdx * KVtileDim * gridDim.y) +
            (1 * KVtileDim) * blockIdx.y;

        const uint cdWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
        const uint cdWarpTileXEnd = CEIL_DIV(KVtileDim, blockDim.x);
        for (uint cdWarpTileYIdx = 0; cdWarpTileYIdx < cdWarpTileYEnd;
             ++cdWarpTileYIdx) {
          for (uint cdWarpTileXIdx = 0; cdWarpTileXIdx < cdWarpTileXEnd;
               ++cdWarpTileXIdx) {
            //? cTileIdxes
            //* shared memory:
            const uint cdWarpTileThreadSharedMemYIdx =
                blockDim.y * cdWarpTileYIdx + threadIdx.y;
            const uint cdWarpTileThreadSharedMemXIdx =
                blockDim.x * cdWarpTileXIdx + threadIdx.x;
            //* global memory:
            const uint cdWarpTileBlockGlobalMemIdx =
                cdTileBlockGlobalMemIdx +
                (seqLen * blockDim.y) * cdWarpTileYIdx +
                blockDim.x * cdWarpTileXIdx;
            const uint cdWarpTileThreadGlobalMemIdx =
                cdWarpTileBlockGlobalMemIdx + seqLen * threadIdx.y +
                threadIdx.x;

            matC[cdWarpTileThreadGlobalMemIdx] =
                SMEMARRAY(dDTile, KVtileDim, cdWarpTileThreadSharedMemYIdx,
                          cdWarpTileThreadSharedMemXIdx);
          }
        }
#endif

        //! Compute pTile = deltaCTile * D'Tile
        //! Compute rTile = sTile * D'Tile
        // left upper corner of cWarpTileBlock in C (global memory)
        const uint prWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
        const uint prWarpTileXEnd = CEIL_DIV(KVtileDim, blockDim.x);
        for (uint prWarpTileYIdx = 0; prWarpTileYIdx < prWarpTileYEnd;
             ++prWarpTileYIdx) {
          for (uint prWarpTileXIdx = 0; prWarpTileXIdx < prWarpTileXEnd;
               ++prWarpTileXIdx) {
            //? cTileIdxes
            //* shared memory:
            const uint prWarpTileThreadSharedMemYIdx =
                blockDim.y * prWarpTileYIdx + threadIdx.y;
            const uint prWarpTileThreadSharedMemXIdx =
                blockDim.x * prWarpTileXIdx + threadIdx.x;

            // loading from shared memory
            scalar_t deltaC_val =
                SMEMARRAY(dCDcsTile, KVtileDim, prWarpTileThreadSharedMemYIdx,
                          prWarpTileThreadSharedMemXIdx);
            scalar_t dstr_val =
                SMEMARRAY(dstrRTile, KVtileDim, prWarpTileThreadSharedMemYIdx,
                          prWarpTileThreadSharedMemXIdx);
            scalar_t s_val =
                SMEMARRAY(sPTile, KVtileDim, prWarpTileThreadSharedMemYIdx,
                          prWarpTileThreadSharedMemXIdx);

            // pointwise operations
            scalar_t p_val = mul_g(deltaC_val, dstr_val);
            scalar_t r_val = mul_g(s_val, dstr_val);

            // store in shared memory
            SMEMARRAY(sPTile, KVtileDim, prWarpTileThreadSharedMemYIdx,
                      prWarpTileThreadSharedMemXIdx) = p_val;
            SMEMARRAY(dstrRTile, KVtileDim, prWarpTileThreadSharedMemYIdx,
                      prWarpTileThreadSharedMemXIdx) = r_val;
          }
        }

#ifdef OUTPUTPRTile
        //! DEBUG: write P or R Tile to global memory
        // left upper corner of cWarpTileBlock in C (global memory)
        //* cdTile Global Memory Index (Debug only)
        const uint cdTileGridXYGlobalMemIdx =
            batchHeadGridXGlobalMemIdxCD + (seqLen * QtileDim) * qTileIdx;
        const uint cdTileBlockGlobalMemIdx =
            cdTileGridXYGlobalMemIdx + (kvTileIdx * KVtileDim * gridDim.y) +
            (1 * KVtileDim) * blockIdx.y;

        const uint cdWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
        const uint cdWarpTileXEnd = CEIL_DIV(KVtileDim, blockDim.x);
        for (uint cdWarpTileYIdx = 0; cdWarpTileYIdx < cdWarpTileYEnd;
             ++cdWarpTileYIdx) {
          for (uint cdWarpTileXIdx = 0; cdWarpTileXIdx < cdWarpTileXEnd;
               ++cdWarpTileXIdx) {
            //? cTileIdxes
            //* shared memory:
            const uint cdWarpTileThreadSharedMemYIdx =
                blockDim.y * cdWarpTileYIdx + threadIdx.y;
            const uint cdWarpTileThreadSharedMemXIdx =
                blockDim.x * cdWarpTileXIdx + threadIdx.x;
            //* global memory:
            const uint cdWarpTileBlockGlobalMemIdx =
                cdTileBlockGlobalMemIdx +
                (seqLen * blockDim.y) * cdWarpTileYIdx +
                blockDim.x * cdWarpTileXIdx;
            const uint cdWarpTileThreadGlobalMemIdx =
                cdWarpTileBlockGlobalMemIdx + seqLen * threadIdx.y +
                threadIdx.x;
#ifdef OUTPUTPRTileR
            matC[cdWarpTileThreadGlobalMemIdx] =
                SMEMARRAY(dstrRTile, KVtileDim, cdWarpTileThreadSharedMemYIdx,
                          cdWarpTileThreadSharedMemXIdx);
#else
            matC[cdWarpTileThreadGlobalMemIdx] =
                SMEMARRAY(sPTile, KVtileDim, cdWarpTileThreadSharedMemYIdx,
                          cdWarpTileThreadSharedMemXIdx);
#endif
          }
        }
        __syncthreads();
#endif

        //! Compute csDTile = cumsum(deltaDtildeTile) (store in dCDcsTile)
        // TODO extend this with sync between thread blocks over
        // TODO fgate delta computation not implemented yet

        //* 1) Init deltaDcsIterHBM to zero in HBM
        // deltaDcsIterHBM: (gridDim.y x QTileDim)
        const uint deltaDcsChunkArrGridXYGlobalMemIdx =
            batchHeadGridXGlobalMemIdxDeltaDcsChunkArr + (QtileDim)*blockIdx.y;
        // we flatten the threads to 1D along the y-dim (qTileDim)
        // in this way we make sure that global mem access is coalesced
        const uint deltaDcsChunkArrQTileDirEnd =
            CEIL_DIV(QtileDim, blockDim.x * blockDim.y);
        for (uint deltaDcsChunkArrQTileDirIdx = 0;
             deltaDcsChunkArrQTileDirIdx < deltaDcsChunkArrQTileDirEnd;
             ++deltaDcsChunkArrQTileDirIdx) {
          //? deltaDcsIterHBM idxes
          //* TB local idx
          const uint deltaDcsChunkArrQTileDirThreadSharedMemIdx =
              flatThreadIdx +
              blockDim.x * blockDim.y * deltaDcsChunkArrQTileDirIdx;
          //* global memory
          const uint deltaDcsIterHBMXYQtileGlobalMemIdx =
              deltaDcsChunkArrGridXYGlobalMemIdx +
              deltaDcsChunkArrQTileDirThreadSharedMemIdx;

          if (deltaDcsChunkArrQTileDirThreadSharedMemIdx < QtileDim) {
            // set to 0.0f
            csDeltaDTildeChunkArr[deltaDcsIterHBMXYQtileGlobalMemIdx] =
                dscalar_zero<float>();
          }
        }
        __syncthreads();

        //* 1a) Calculate local cumsum along the j-direction (kvTileDim / x-dim)
        //* 1b) If last cumsum tile col is at TB boundary (not at main
        //* diagonal!), write to deltaDcsIterHBM[gridDim.y], sync TBs
        // loop in i-direction (qTileDim / y-dim)
        const uint csDTileYdimEnd = CEIL_DIV(QtileDim, blockDim.x * blockDim.y);
        for (uint csDtileYdimThreadSharedMemIdx = 0;
             csDtileYdimThreadSharedMemIdx < csDTileYdimEnd;
             ++csDtileYdimThreadSharedMemIdx) {
          //? csDTile idxes
          //* shared memory
          const uint csDTileYdimThreadSharedMemIdx =
              flatThreadIdx +
              blockDim.x * blockDim.y * csDtileYdimThreadSharedMemIdx;

          //* csDTile global index (y-dim / QtileDim) (virtual, as never
          // materialized fully)
          const uint csDTileYdimThreadIdx =
              sTileYdimBlockYIdx + csDTileYdimThreadSharedMemIdx;

          if (csDTileYdimThreadSharedMemIdx < QtileDim) {
            float acc = 0.0f;
            bool reachedMainDiagonal = false;
            for (uint csDTileXdimThreadSharedMemIdx = 0;
                 csDTileXdimThreadSharedMemIdx < KVtileDim;
                 ++csDTileXdimThreadSharedMemIdx) {
              //* csDtile global index (x-dim / KVtiledim) (virtual, as never
              // materialized fully)
              const uint csDTileXdimThreadIdx =
                  sTileXdimBlockYIdx + csDTileXdimThreadSharedMemIdx;
              scalar_t dcs_val = dscalar_zero<scalar_t>();
              if (csDTileYdimThreadIdx > csDTileXdimThreadIdx) {
                scalar_t d_val =
                    SMEMARRAY(dDTile, KVtileDim, csDTileYdimThreadSharedMemIdx,
                              csDTileXdimThreadSharedMemIdx);
                acc = add_g(acc, type2float(d_val));
                dcs_val = float2type<scalar_t>(acc);
              } else {
                reachedMainDiagonal = true;
              }
              SMEMARRAY(dCDcsTile, KVtileDim, csDTileYdimThreadSharedMemIdx,
                        csDTileXdimThreadSharedMemIdx) = dcs_val;
            }
            if (!reachedMainDiagonal) {
              // write to global memory
              const uint deltaDcsChunkArrThreadGlobalMemIdx =
                  deltaDcsChunkArrGridXYGlobalMemIdx +
                  csDTileYdimThreadSharedMemIdx;
              csDeltaDTildeChunkArr[deltaDcsChunkArrThreadGlobalMemIdx] = acc;
            }
          }
        }
        __syncthreads();
        gridGroup.sync();

        //* 2) Build the cumsum correction per TB: deltaDcsLoopHBM +
        // sum(deltaDcsIterHBM[<blockIdx.y])
        // store the result in mDeltaDcsCorrChunk in SRAM
        // loop in i-direction (qTileDim / y-dim) with flattened threads
        for (uint csDtileYdimThreadSharedMemIdx = 0;
             csDtileYdimThreadSharedMemIdx < csDTileYdimEnd;
             ++csDtileYdimThreadSharedMemIdx) {
          //? csDTile idxes
          //* shared memory
          const uint csDTileYdimThreadSharedMemIdx =
              flatThreadIdx +
              blockDim.x * blockDim.y * csDtileYdimThreadSharedMemIdx;

          if (csDTileYdimThreadSharedMemIdx < QtileDim) {
            // global memory index for the cumsum correction
            const uint csDeltaDTildeVecThreadGlobalMemIdx =
                nmfChunkBlockGlobalMemIdx + csDTileYdimThreadSharedMemIdx;
            // load the global cumsum correction from HBM
            // and init the total cumusum correction in mDeltaDcsCorrChunk
            float total_cumsum_corr =
                csDeltaDTildeVec[csDeltaDTildeVecThreadGlobalMemIdx];

            // loop over gridDim.y and sum up the cumsum corrections
            // of the other thread blocks with lower blockIdx.y
            for (uint blockIdxY = 0; blockIdxY < blockIdx.y; ++blockIdxY) {
              const uint deltaDcsChunkArrThreadGlobalMemIdx =
                  batchHeadGridXGlobalMemIdxDeltaDcsChunkArr +
                  QtileDim * blockIdxY + csDTileYdimThreadSharedMemIdx;
              total_cumsum_corr = add_g(
                  total_cumsum_corr,
                  csDeltaDTildeChunkArr[deltaDcsChunkArrThreadGlobalMemIdx]);

#ifdef DEBUG_DeltaDCS0
              if ((blockIdx.x == 0) && (blockIdx.y == 1) && (iIdx <= jIdx) &&
                  (flatThreadIdx <= 8)) {
                printf("blockIdx(x,y)=(%d,%d), FTIdx=%d, ijIdx(i,j)=(%d,%d), "
                       "sTileXYIdx(x,y)=(%d,%d), csDYdimSMIdx=%d, bIdxY=%d, "
                       "tot_cs_cor=%f\n",
                       blockIdx.x, blockIdx.y, flatThreadIdx, iIdx, jIdx,
                       sTileXdimBlockYIdx, sTileYdimBlockYIdx,
                       csDTileYdimThreadSharedMemIdx, blockIdxY,
                       total_cumsum_corr);
              }
#endif
            }
            // store the total cumsum correction in mDeltaDcsCorrChunk
            SMEMVECTOR(mDeltaDcsCorrChunk, csDTileYdimThreadSharedMemIdx) =
                float2type<scalar_t>(total_cumsum_corr);

#ifdef DEBUG_DeltaDCS1
            if ((blockIdx.x == 0) && (blockIdx.y <= 1) && (jIdx == 0) &&
                (flatThreadIdx <= 8)) {
              printf("blockIdx(x,y)=(%d,%d), FTIdx=%d, ijIdx(i,j)=(%d,%d), "
                     "sTileXYIdx(x,y)=(%d,%d), csDYdimSMIdx=%d, "
                     "tot_cs_cor=%f\n",
                     blockIdx.x, blockIdx.y, flatThreadIdx, iIdx, jIdx,
                     sTileXdimBlockYIdx, sTileYdimBlockYIdx,
                     csDTileYdimThreadSharedMemIdx, total_cumsum_corr);
            }
#endif
          }
        }

        //* 3a) Add cumsum correction to local cumsum in DcsTile
        //* 3b) Store the cumsum result of the last col of the TB closest to
        //*     the main diagonal (but not at the main diagonal!) in
        //*     deltaDcsLoopHBM
        // outer loop: in j-direction (qTileDim / y-dim) with flattened threads
        // inner loop: in i-direction (kvTileDim / x-dim) per thread
        for (uint csDtileYdimThreadSharedMemIdx = 0;
             csDtileYdimThreadSharedMemIdx < csDTileYdimEnd;
             ++csDtileYdimThreadSharedMemIdx) {
          //? csDTile idxes
          //* shared memory
          const uint csDTileYdimThreadSharedMemIdx =
              flatThreadIdx +
              blockDim.x * blockDim.y * csDtileYdimThreadSharedMemIdx;
          //* csDTile global index (y-dim / QtileDim) (virtual, as never
          // materialized fully)
          const uint csDTileYdimThreadIdx =
              sTileYdimBlockYIdx + csDTileYdimThreadSharedMemIdx;

          if (csDTileYdimThreadSharedMemIdx < QtileDim) {
            const uint csDeltaDTildeVecThreadGlobalMemIdx =
                nmfChunkBlockGlobalMemIdx + csDTileYdimThreadSharedMemIdx;

            // load the total cumsum correction from SRAM
            scalar_t total_cumsum_corr =
                SMEMVECTOR(mDeltaDcsCorrChunk, csDTileYdimThreadSharedMemIdx);

            bool reachedMainDiagonal = false;
            scalar_t dcs_corr_val = dscalar_zero<scalar_t>();

            for (uint csDTileXdimThreadSharedMemIdx = 0;
                 csDTileXdimThreadSharedMemIdx < KVtileDim;
                 ++csDTileXdimThreadSharedMemIdx) {

              //* csDtile global index (x-dim / KVtiledim) (virtual, as never
              // materialized fully)
              const uint csDTileXdimThreadIdx =
                  sTileXdimBlockYIdx + csDTileXdimThreadSharedMemIdx;

              if (csDTileYdimThreadIdx > csDTileXdimThreadIdx) {
                // below main diagonal
                // load&update the cumsum(Dtilde) val
                dcs_corr_val = SMEMARRAY(dCDcsTile, KVtileDim,
                                         csDTileYdimThreadSharedMemIdx,
                                         csDTileXdimThreadSharedMemIdx);

                dcs_corr_val = add_g(dcs_corr_val, total_cumsum_corr);

                // store the updated cumsum(Dtilde) val
                SMEMARRAY(dCDcsTile, KVtileDim, csDTileYdimThreadSharedMemIdx,
                          csDTileXdimThreadSharedMemIdx) = dcs_corr_val;
              } else {
                reachedMainDiagonal = true;
                break;
              }
            }
#ifdef DEBUG_DeltaDCS2
            if ((blockIdx.x == 0) && (blockIdx.y == gridDim.y - 1) &&
                (iIdx >= jIdx) && (flatThreadIdx <= 8)) {
              printf("blockIdx(x,y)=(%d,%d), FTIdx=%d, ijIdx(i,j)=(%d,%d), "
                     "sTileXYIdx(x,y)=(%d,%d), csDYdimGBIdx=%d, "
                     "dcs_cv=%f, reachedMD=%d\n",
                     blockIdx.x, blockIdx.y, flatThreadIdx, iIdx, jIdx,
                     sTileXdimBlockYIdx, sTileYdimBlockYIdx,
                     csDTileYdimThreadIdx, type2float(dcs_corr_val),
                     reachedMainDiagonal);
            }
#endif
            if ((!reachedMainDiagonal) && (iIdx >= jIdx) &&
                (blockIdx.y == gridDim.y - 1)) {
              // onle the thread block closest to the main diagonal
              // but not at the main diagonal should write to global memory
              csDeltaDTildeVec[csDeltaDTildeVecThreadGlobalMemIdx] =
                  type2float(dcs_corr_val);
            }
          }
        }

#ifdef OUTPUTDcsTile
        //! DEBUG: write cumsum(Dtilde) Tile to global memory
        // left upper corner of cWarpTileBlock in C (global memory)
        //* cdTile Global Memory Index (Debug only)
        const uint cdTileGridXYGlobalMemIdx =
            batchHeadGridXGlobalMemIdxCD + (seqLen * QtileDim) * qTileIdx;
        const uint cdTileBlockGlobalMemIdx =
            cdTileGridXYGlobalMemIdx + (kvTileIdx * KVtileDim * gridDim.y) +
            (1 * KVtileDim) * blockIdx.y;

        const uint cdWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y);
        const uint cdWarpTileXEnd = CEIL_DIV(KVtileDim, blockDim.x);
        for (uint cdWarpTileYIdx = 0; cdWarpTileYIdx < cdWarpTileYEnd;
             ++cdWarpTileYIdx) {
          for (uint cdWarpTileXIdx = 0; cdWarpTileXIdx < cdWarpTileXEnd;
               ++cdWarpTileXIdx) {
            //? cTileIdxes
            //* shared memory:
            const uint cdWarpTileThreadSharedMemYIdx =
                blockDim.y * cdWarpTileYIdx + threadIdx.y;
            const uint cdWarpTileThreadSharedMemXIdx =
                blockDim.x * cdWarpTileXIdx + threadIdx.x;
            //* global memory:
            const uint cdWarpTileBlockGlobalMemIdx =
                cdTileBlockGlobalMemIdx +
                (seqLen * blockDim.y) * cdWarpTileYIdx +
                blockDim.x * cdWarpTileXIdx;
            const uint cdWarpTileThreadGlobalMemIdx =
                cdWarpTileBlockGlobalMemIdx + seqLen * threadIdx.y +
                threadIdx.x;

            matC[cdWarpTileThreadGlobalMemIdx] =
                SMEMARRAY(dCDcsTile, KVtileDim, cdWarpTileThreadSharedMemYIdx,
                          cdWarpTileThreadSharedMemXIdx);
          }
        }
#endif

        //! sum up deltaIChunk & deltaFChunk and update in SRAM
        // sum along i-direction (qTileDim / y-dim)
        // loop in j-direction (kvTileDim / x-dim)
        const uint csDtileXdimXdimEnd =
            CEIL_DIV(KVtileDim, blockDim.x * blockDim.y);
        for (uint csDtileXdimThreadSharedMemIdx = 0;
             csDtileXdimThreadSharedMemIdx < csDtileXdimXdimEnd;
             ++csDtileXdimThreadSharedMemIdx) {
          //? dIdFChunk idxes
          //* shared memory
          const uint dIdFChunkXdimThreadSharedMemIdx =
              flatThreadIdx +
              blockDim.x * blockDim.y * csDtileXdimThreadSharedMemIdx;

          //* dTile global index (x-dim / kvTileDim) (virtual, as never
          // materialized fully)
          const uint dTileXdimThreadIdx =
              sTileXdimBlockYIdx + dIdFChunkXdimThreadSharedMemIdx;

          if (dIdFChunkXdimThreadSharedMemIdx < KVtileDim) {
            float acc_deltaI = 0.0f;
            float acc_deltaF = 0.0f;
            for (uint csDtileYdimThreadSharedMemIdx = 0;
                 csDtileYdimThreadSharedMemIdx < QtileDim;
                 ++csDtileYdimThreadSharedMemIdx) {

              //* dTile global index (y-dim / QtileDim) (virtual, as never
              // materialized fully)
              const uint dTileYdimThreadIdx =
                  sTileYdimBlockYIdx + csDtileYdimThreadSharedMemIdx;
#ifdef DEBUG_deltaISUM0
              if ((blockIdx.x == 0) && (blockIdx.y == 0) &&
                  (flatThreadIdx < 1)) {
                printf("!qTileIdx=%d, kvTileIdx=%d, dTileXdimThreadIdx=%d, "
                       "dIdFChunkXdimTSMIdx=%d, flatTidx=%d, "
                       "tbIdxXY=(%d,%d): csDtileIdx=%d, dTileXdimTIdx=%d, "
                       "dTileYdimTIdx=%d\n",
                       qTileIdx, kvTileIdx, dTileXdimThreadIdx,
                       dIdFChunkXdimThreadSharedMemIdx, flatThreadIdx,
                       threadIdx.x, threadIdx.y, csDtileYdimThreadSharedMemIdx,
                       dTileXdimThreadIdx, dTileYdimThreadIdx);
              }
#endif
              // sum up deltaIChunk
              if (dTileYdimThreadIdx >= dTileXdimThreadIdx) {
                //? sum the entries in deltaDtildeTile
                scalar_t deltaI_val =
                    SMEMARRAY(dDTile, KVtileDim, csDtileYdimThreadSharedMemIdx,
                              dIdFChunkXdimThreadSharedMemIdx);
                acc_deltaI = add_g(acc_deltaI, type2float(deltaI_val));
#ifdef DEBUG_deltaISUM1
                if ((blockIdx.x == 0) && (blockIdx.y == 0) &&
                    (flatThreadIdx < 1)) {
                  printf("qTileIdx=%d, kvTileIdx=%d, dTileXdimThreadIdx=%d, "
                         "dIdFChunkXdimTSMIdx=%d, flatTidx=%d, "
                         "tbIdxXY=(%d,%d), csDtileIdx=%d: "
                         "acc_deltaI=%f\n",
                         qTileIdx, kvTileIdx, dTileXdimThreadIdx,
                         dIdFChunkXdimThreadSharedMemIdx, flatThreadIdx,
                         threadIdx.x, threadIdx.y,
                         csDtileYdimThreadSharedMemIdx, type2float(deltaI_val));
                }
#endif
              }

              // sum up deltaFChunk
              if (dTileYdimThreadIdx > dTileXdimThreadIdx) {
                //? sum the entries in DcsTile
                scalar_t deltaF_val = SMEMARRAY(
                    dCDcsTile, KVtileDim, csDtileYdimThreadSharedMemIdx,
                    dIdFChunkXdimThreadSharedMemIdx);
                acc_deltaF = add_g(acc_deltaF, type2float(deltaF_val));
              }

            } // end for (csDtileYdimThreadSharedMemIdx)

#ifdef DEBUG_deltaISUM2
            if ((blockIdx.x == 0) && (blockIdx.y == 0) && (flatThreadIdx < 1)) {
              printf("qTileIdx=%d, kvTileIdx=%d, dTileXdimThreadIdx=%d, "
                     "dIdFChunkXdimThreadSharedMemIdx=%d, flatTidx=%d, "
                     "tbIdxXY=(%d,%d): "
                     "acc_deltaI=%f\n",
                     qTileIdx, kvTileIdx, dTileXdimThreadIdx,
                     dIdFChunkXdimThreadSharedMemIdx, flatThreadIdx,
                     threadIdx.x, threadIdx.y, acc_deltaI);
            }
#endif

            // update deltaIChunk & deltaFChunk in SMEM
            scalar_t deltaI_val =
                SMEMVECTOR(deltaIChunk, dIdFChunkXdimThreadSharedMemIdx);
            SMEMVECTOR(deltaIChunk, dIdFChunkXdimThreadSharedMemIdx) =
                float2type<scalar_t>(add_g(type2float(deltaI_val), acc_deltaI));

            scalar_t deltaFbar_val =
                SMEMVECTOR(deltaFChunk, dIdFChunkXdimThreadSharedMemIdx);
            SMEMVECTOR(deltaFChunk, dIdFChunkXdimThreadSharedMemIdx) =
                float2type<scalar_t>(
                    add_g(type2float(deltaFbar_val), acc_deltaF));
          } // end if (dIdFChunkXdimThreadSharedMemIdx < KVtileDim)
        }   // end for (csDtileXdimThreadSharedMemIdx)

        //! Compute deltaQTile = pTile  (kTile/sqrt(d))
        // (QtileDim x dimHeads) = (QtileDim x KVtileDim) x (KVtileDim x
        // dimHeads)
        // loops over deltaQTile rows (outer) and columns (inner)
        const uint deltaQWarpTileYEnd = CEIL_DIV(QtileDim, blockDim.y); // rows
        const uint deltaQWarpTileXEnd = CEIL_DIV(dimHeads, blockDim.x); // cols
        for (uint deltaQWarpTileYIdx = 0;
             deltaQWarpTileYIdx < deltaQWarpTileYEnd; ++deltaQWarpTileYIdx) {
          for (uint deltaQWarpTileXIdx = 0;
               deltaQWarpTileXIdx < deltaQWarpTileXEnd; ++deltaQWarpTileXIdx) {
            //? cTileIdxes
            //* shared memory:
            const uint deltaQWarpTileThreadSharedMemYIdx =
                blockDim.y * deltaQWarpTileYIdx + threadIdx.y;
            const uint deltaQWarpTileThreadSharedMemXIdx =
                blockDim.x * deltaQWarpTileXIdx + threadIdx.x;

            // scalar_t qk_acc = dscalar_zero<scalar_t>();
            float acc = 0.0f;
            for (uint i = 0; i < KVtileDim; ++i) {
              acc = add_g(acc,
                          type2float(mul_g(
                              SMEMARRAY(sPTile, KVtileDim,
                                        deltaQWarpTileThreadSharedMemYIdx, i),
                              SMEMARRAY(kTile, dimHeads, i,
                                        deltaQWarpTileThreadSharedMemXIdx))));
            }

            // compute deltaQTile
            scalar_t deltaQ_val =
                float2type<scalar_t>(mul_g(acc, rsqrtf(type2float(dimHeads))));
            SMEMARRAY(deltaQTile, dimHeads, deltaQWarpTileThreadSharedMemYIdx,
                      deltaQWarpTileThreadSharedMemXIdx) = deltaQ_val;
          }
        }
        __syncthreads();

        gridGroup.sync();

        //! Atomic add deltaQTile to deltaQ in HBM
        // We sum up the deltaQTiles in the different thread blocks in the
        // global memory loops over rows (outer) and columns (inner) of
        // deltaQTile
        for (uint deltaQWarpTileYIdx = 0;
             deltaQWarpTileYIdx < deltaQWarpTileYEnd; ++deltaQWarpTileYIdx) {
          for (uint deltaQWarpTileXIdx = 0;
               deltaQWarpTileXIdx < deltaQWarpTileXEnd; ++deltaQWarpTileXIdx) {

            //? deltaQWarpTileIdxes for deltaQ-tile
            //* shared memory:
            const uint deltaQWarpTileThreadSharedMemYIdx =
                blockDim.y * deltaQWarpTileYIdx + threadIdx.y;
            const uint deltaQWarpTileThreadSharedMemXIdx =
                blockDim.x * deltaQWarpTileXIdx + threadIdx.x;

            //* global memory:
            // left upper corner of qTileBlock in Q (global memory)
            const uint deltaQWarpTileBlockGlobalMemIdx =
                qdHdQTileBlockGlobalMemIdx +
                (dimHeads * blockDim.y) * deltaQWarpTileYIdx +
                blockDim.x * deltaQWarpTileXIdx;
            const uint deltaQWarpTileThreadGlobalMemIdx =
                deltaQWarpTileBlockGlobalMemIdx + dimHeads * threadIdx.y +
                threadIdx.x;

            // load deltaQ val from shared memory
            scalar_t deltaQ_val = SMEMARRAY(deltaQTile, dimHeads,
                                            deltaQWarpTileThreadSharedMemYIdx,
                                            deltaQWarpTileThreadSharedMemXIdx);

            // atomic add to deltaQ in HBM
            atomicAdd(&(deltaQ[deltaQWarpTileThreadGlobalMemIdx]), deltaQ_val);
          }
        }
        __syncthreads();

        //! Compute deltaKTile = pTile^T  (qTile/sqrt(d)) and update in SRAM
        // (KVtileDim x dimHeads) = (KVtileDim x QtileDim) x (QtileDim x
        // dimHeads)
        // loops over deltaKTile rows (outer) and columns (inner)
        const uint deltaKWarpTileYEnd = CEIL_DIV(KVtileDim, blockDim.y); // rows
        const uint deltaKWarpTileXEnd = CEIL_DIV(dimHeads, blockDim.x);  // cols
        for (uint deltaKWarpTileYIdx = 0;
             deltaKWarpTileYIdx < deltaKWarpTileYEnd; ++deltaKWarpTileYIdx) {
          for (uint deltaKWarpTileXIdx = 0;
               deltaKWarpTileXIdx < deltaKWarpTileXEnd; ++deltaKWarpTileXIdx) {
            //? cTileIdxes
            //* shared memory:
            const uint deltaKWarpTileThreadSharedMemYIdx =
                blockDim.y * deltaKWarpTileYIdx + threadIdx.y;
            const uint deltaKWarpTileThreadSharedMemXIdx =
                blockDim.x * deltaKWarpTileXIdx + threadIdx.x;

            // scalar_t qk_acc = dscalar_zero<scalar_t>();
            float acc = 0.0f;
            for (uint i = 0; i < QtileDim; ++i) {
              acc = add_g(acc,
                          type2float(mul_g(
                              SMEMARRAY(sPTile, KVtileDim, i,
                                        deltaKWarpTileThreadSharedMemYIdx),
                              SMEMARRAY(qTile, dimHeads, i,
                                        deltaKWarpTileThreadSharedMemXIdx))));
            }

            // compute deltaKTile
            scalar_t deltaK_val_new =
                float2type<scalar_t>(mul_g(acc, rsqrtf(type2float(dimHeads))));

            // update deltaKTile in shared memory
            scalar_t deltaK_val_old = SMEMARRAY(
                deltaKTile, dimHeads, deltaKWarpTileThreadSharedMemYIdx,
                deltaKWarpTileThreadSharedMemXIdx);

            SMEMARRAY(deltaKTile, dimHeads, deltaKWarpTileThreadSharedMemYIdx,
                      deltaKWarpTileThreadSharedMemXIdx) =
                add_g(deltaK_val_old, deltaK_val_new);
          }
        }
        __syncthreads();

        gridGroup.sync();

        //! Compute deltaVTile = rTile^T  (deltaHTile * 1 / nChunk) and update
        //! in SRAM
        // (KVtileDim x dimHeads) = (KVtileDim x QtileDim) x (QtileDim x
        // dimHeads)
        // loops over deltaKTile rows (outer) and columns (inner)
        const uint deltaVWarpTileYEnd = CEIL_DIV(KVtileDim, blockDim.y); // rows
        const uint deltaVWarpTileXEnd = CEIL_DIV(dimHeads, blockDim.x);  // cols
        for (uint deltaVWarpTileYIdx = 0;
             deltaVWarpTileYIdx < deltaVWarpTileYEnd; ++deltaVWarpTileYIdx) {
          for (uint deltaVWarpTileXIdx = 0;
               deltaVWarpTileXIdx < deltaVWarpTileXEnd; ++deltaVWarpTileXIdx) {
            //? deltaVTileIdxes
            //* shared memory:
            const uint deltaVWarpTileThreadSharedMemYIdx =
                blockDim.y * deltaVWarpTileYIdx + threadIdx.y;
            const uint deltaVWarpTileThreadSharedMemXIdx =
                blockDim.x * deltaVWarpTileXIdx + threadIdx.x;

            float acc = 0.0f;
            for (uint i = 0; i < QtileDim; ++i) {
              scalar_t r_val = SMEMARRAY(dstrRTile, KVtileDim, i,
                                         deltaVWarpTileThreadSharedMemYIdx);

              scalar_t deltaH_val = SMEMARRAY(
                  deltaHTile, dimHeads, i, deltaVWarpTileThreadSharedMemXIdx);
              scalar_t n_val = SMEMVECTOR(nChunk, i);

              deltaH_val = div_g(deltaH_val, n_val);

              acc = add_g(acc, type2float(mul_g(r_val, deltaH_val)));
            }

            // compute deltaVTile
            scalar_t deltaV_val_new = float2type<scalar_t>(acc);

            // update deltaVTile in shared memory
            scalar_t deltaV_val_old = SMEMARRAY(
                deltaVTile, dimHeads, deltaVWarpTileThreadSharedMemYIdx,
                deltaVWarpTileThreadSharedMemXIdx);

            SMEMARRAY(deltaVTile, dimHeads, deltaVWarpTileThreadSharedMemYIdx,
                      deltaVWarpTileThreadSharedMemXIdx) =
                add_g(deltaV_val_old, deltaV_val_new);
          }
        }
        __syncthreads();

        gridGroup.sync();

      } // end looplevel 2 (i-loop)

      gridGroup.sync();

      //! Store deltaKTile & deltaVTile in HBM
      // loops over rows (outer) and columns (inner) of kTile and vTile
      const uint dKdVWarpTileYEnd = CEIL_DIV(KVtileDim, blockDim.y); // rows
      const uint dKdVWarpTileXEnd = CEIL_DIV(dimHeads, blockDim.x);  // cols
      for (uint dKdVWarpTileYIdx = 0; dKdVWarpTileYIdx < dKdVWarpTileYEnd;
           ++dKdVWarpTileYIdx) {
        for (uint dKdVWarpTileXIdx = 0; dKdVWarpTileXIdx < dKdVWarpTileXEnd;
             ++dKdVWarpTileXIdx) {
          //? dKdVWarpTileIdxes for k-tile AND v-tile
          //* shared memory:
          const uint dKdVWarpTileThreadSharedMemYIdx =
              blockDim.y * dKdVWarpTileYIdx + threadIdx.y;
          const uint dKdVWarpTileThreadSharedMemXIdx =
              blockDim.x * dKdVWarpTileXIdx + threadIdx.x;
          //* global memory:
          // left upper corner of kTileBlock in K (global memory)
          const uint kvWarpTileBlockGlobalMemIdx =
              kvdKdVTileBlockGlobalMemIdx +
              (dimHeads * blockDim.y) * dKdVWarpTileYIdx +
              blockDim.x * dKdVWarpTileXIdx;
          const uint kvWarpTileThreadGlobalMemIdx =
              kvWarpTileBlockGlobalMemIdx + dimHeads * threadIdx.y +
              threadIdx.x;

          // write to HBM
          deltaK[kvWarpTileThreadGlobalMemIdx] =
              SMEMARRAY(deltaKTile, dimHeads, dKdVWarpTileThreadSharedMemYIdx,
                        dKdVWarpTileThreadSharedMemXIdx);
          deltaV[kvWarpTileThreadGlobalMemIdx] =
              SMEMARRAY(deltaVTile, dimHeads, dKdVWarpTileThreadSharedMemYIdx,
                        dKdVWarpTileThreadSharedMemXIdx);

          // clear SRAM for next iteration
          SMEMARRAY(deltaKTile, dimHeads, dKdVWarpTileThreadSharedMemYIdx,
                    dKdVWarpTileThreadSharedMemXIdx) = dscalar_zero<scalar_t>();
          SMEMARRAY(deltaVTile, dimHeads, dKdVWarpTileThreadSharedMemYIdx,
                    dKdVWarpTileThreadSharedMemXIdx) = dscalar_zero<scalar_t>();
        }
      }
      __syncthreads();

      //! Store deltaIChunk & deltaFChunk in HBM
      // loop in j-direction (kvTileDim / x-dim)
      const uint dIdFChunkEnd = CEIL_DIV(KVtileDim, blockDim.x * blockDim.y);
      for (uint dIdFChunkIdx = 0; dIdFChunkIdx < dIdFChunkEnd; ++dIdFChunkIdx) {
        //? dIdFChunk idxes
        //* shared memory
        const uint dIdFChunkThreadSharedMemIdx =
            flatThreadIdx + blockDim.x * blockDim.y * dIdFChunkIdx;
        //* global memory
        const uint dIdFThreadGlobalMemIdx =
            iChunkBlockGlobalMemIdx + flatThreadIdx;

        //* global dFIdx (virtual, as never materialized fully)
        const uint dFChunkXdimBlockYThreadIdx =
            sTileXdimBlockYIdx + dIdFChunkThreadSharedMemIdx;

        if (dIdFChunkThreadSharedMemIdx < KVtileDim) {
          deltaIGatePreact[dIdFThreadGlobalMemIdx] =
              SMEMVECTOR(deltaIChunk, dIdFChunkThreadSharedMemIdx);

          // multiply with sigmoid derivative: sigmoid(-fGatePreact)
          scalar_t deltaFbar_val =
              SMEMVECTOR(deltaFChunk, dIdFChunkThreadSharedMemIdx);

          // avoid accessing out of bounds (need to check that
          // the very last value is not written and not accessed)
          if (dFChunkXdimBlockYThreadIdx < seqLen - 1) {
            // need to shift index since the first forgetgate f_1 is not used
            scalar_t fPreact_val = fGatePreact[dIdFThreadGlobalMemIdx + 1];

            scalar_t deltaF_val =
                mul_g(deltaFbar_val, sigmoid_g(neg_g(fPreact_val)));

            deltaFGatePreact[dIdFThreadGlobalMemIdx + 1] = deltaF_val;
          }
#ifdef DEBUG_WRdeltaI
          if ((blockIdx.x == 0) && (blockIdx.y == 0) && (flatThreadIdx <= 8)) {
            printf("kvTileIdx=%d, dIdFChunkIdx=%d"
                   "iChunkBlockGlobalMemIdx=%d, flatTidx=%d, tbIdxXY=(%d,%d): "
                   "deltaI[%d]=%f\n",
                   kvTileIdx, dIdFChunkIdx, iChunkBlockGlobalMemIdx,
                   flatThreadIdx, threadIdx.x, threadIdx.y,
                   dIdFThreadGlobalMemIdx,
                   type2float(
                       SMEMVECTOR(deltaIChunk, dIdFChunkThreadSharedMemIdx)));
          }
#endif
        }
      }
      __syncthreads();

    } // end looplevel 1 (j-loop)

  } // end looplevel 0
} // kernels::vlstm_fw

template <typename scalar_t>
void kernel_dispatchers::vlstm_bw_dispatch(
    scalar_t *deltaQ, scalar_t *deltaK, scalar_t *deltaV,
    scalar_t *deltaIGatePreact, scalar_t *deltaFGatePreact, scalar_t *matC,
    scalar_t *deltaH, scalar_t *matQ, scalar_t *matK, scalar_t *matV,
    scalar_t *iGatePreact, scalar_t *fGatePreact, scalar_t *vecN,
    scalar_t *vecM, int batchSize, int numHeads, int seqLen, int dimHeads) {
  printf("B: %d, NH: %d, S: %d, DH: %d\n", batchSize, numHeads, seqLen,
         dimHeads);
  const int TblockDim = TBLOCK_DIM; // matmul blockdim
  const int QtileDim = QTILE_DIM;   // blockdim for Q along seqLen dim
  const int KVtileDim = KVTILE_DIM; // blockdim for K&V along seqLen dim

  // kernel asserts
  if ((seqLen % QtileDim != 0) || (seqLen % KVtileDim != 0)) {
    printf("seqLen must be divisible by QblockDim and KVblockDim\n");
  }

  // determine the number of blocks and threads
  const dim3 blockDims(TblockDim, TblockDim);

  // TODO: determine gridDims
  // Note @mbeck: should be dynamically allocated.
  // At first parallelize across batchSize and numHeads.
  // If more streaming multiprocessors available, parallelize across seqLen.
  //! NOTE: for now we only parallelize across batchSize and numHeads
  // TODO Need to dynamically check how many blocks we can launch
  // TODO add check if batchSize*numHeads exceeds max gridDim.x

  const uint gridDimY = 2;
  const dim3 gridDims(batchSize * numHeads, gridDimY);
  //   const dim3 gridDims(1, 1);

  //! calculate dynamic shared memory size
  // TODO understand how memory padding works!
  // Why at innermost dim? Because memory is organized consecutively

  //? input-output tiles
  const uint qdQdHTileSharedMemSize =
      sizeof(scalar_t) * QtileDim * (dimHeads + SHARED_MEM_PADDING);
  const uint kvdKdVTileSharedMemSize =
      sizeof(scalar_t) * KVtileDim * (dimHeads + SHARED_MEM_PADDING);

  // See here:
  // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#device-memory-accesses
  // the idea of the padding is that every number is stored in a different
  // memory bank this should help to avoid bank conflicts as many threads need
  // to access the same input and forget gate values at the same time for the
  // gate matrix computation
  // TODO check if this is really helping!
  const uint ididfChunkSharedMemSize =
      sizeof(scalar_t) * KVtileDim * (1 + SHARED_MEM_PADDING);

  const uint nmfChunkSharedMemSize =
      sizeof(scalar_t) * QtileDim * (1 + SHARED_MEM_PADDING);

  //? intermediate tiles
  const uint sdprdcddTileSharedMemSize =
      sizeof(scalar_t) * QtileDim * (KVtileDim + SHARED_MEM_PADDING);

  // we keep these as float as it acts as accumulator
  // for the fTileRow during D' computation
  const uint fTileRowSharedMemSize =
      sizeof(float) * KVtileDim * (1 + SHARED_MEM_PADDING);

  const uint sharedMemorySize =
      3 * qdQdHTileSharedMemSize + 4 * kvdKdVTileSharedMemSize +
      3 * ididfChunkSharedMemSize + 3 * nmfChunkSharedMemSize +
      4 * sdprdcddTileSharedMemSize + 1 * fTileRowSharedMemSize;

  printf("blocksxy: %d-%d, threadsxy: %d-%d, shared_mem in bytes: %d\n",
         gridDims.x, gridDims.y, blockDims.x, blockDims.y, sharedMemorySize);
  // hipSetDevice(0);

  hipStream_t stream;
  hipStreamCreate(&stream);

  // TODO bring this back later. For debugging purposes, we need to allocate
  // memory in torch
  //? Allocate intermediate global memory for cumsum(deltaDtildeTile) along
  // KVdim
  //* csDeltaDTildeChunkArr: Used for sync within all y-dim TBs
  uint csDeltaDTildeChunkArrGlobalMemSize =
      sizeof(float) * batchSize * numHeads * QtileDim * gridDimY;
  float *csDeltaDTildeChunkArr;
  gpuErrchk(hipMalloc((void **)&csDeltaDTildeChunkArr,
                       csDeltaDTildeChunkArrGlobalMemSize));

  //* csDeltaDTildeVec: Used to store previous computations over j - iterations
  //(iterations over the gridDimY)
  uint csDeltaDTildeVecGlobalMemSize =
      sizeof(float) * batchSize * numHeads * seqLen;
  float *csDeltaDTildeVec;
  gpuErrchk(
      hipMalloc((void **)&csDeltaDTildeVec, csDeltaDTildeVecGlobalMemSize));
  // init the memory to zero
  // hipMemset only works with integers, but float 0.0 has 0000 0000 in binary,
  // so it should work
  gpuErrchk(hipMemset(csDeltaDTildeVec, 0, csDeltaDTildeVecGlobalMemSize));

  auto kernel = kernels::vlstm_bw<scalar_t, TblockDim, QtileDim, KVtileDim>;
  hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributePreferredSharedMemoryCarveout,
                       cudaSharedmemCarveoutMaxShared);
  hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize,
                       sharedMemorySize);
  // define void* pointers to the kernel arguments
  // TODO adapt this!
  void *kernelArgs[] = {(void *)&deltaQ,
                        (void *)&deltaK,
                        (void *)&deltaV,
                        (void *)&deltaIGatePreact,
                        (void *)&deltaFGatePreact,
                        (void *)&matC,
                        (void *)&deltaH,
                        (void *)&matQ,
                        (void *)&matK,
                        (void *)&matV,
                        (void *)&iGatePreact,
                        (void *)&fGatePreact,
                        (void *)&vecN,
                        (void *)&vecM,
                        (void *)&csDeltaDTildeChunkArr,
                        (void *)&csDeltaDTildeVec,
                        (void *)&batchSize,
                        (void *)&numHeads,
                        (void *)&seqLen,
                        (void *)&dimHeads};

  hipLaunchCooperativeKernel((void *)kernel, gridDims, blockDims, kernelArgs,
                              sharedMemorySize, stream);

  gpuErrchk(hipPeekAtLastError());

  // free the allocated memory
  gpuErrchk(hipFree(csDeltaDTildeChunkArr));
  gpuErrchk(hipFree(csDeltaDTildeVec));

  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());
}

// this is needed to make sure that the compiler instantiates the template
template void kernel_dispatchers::vlstm_bw_dispatch<__hip_bfloat16>(
    __hip_bfloat16 *deltaQ, __hip_bfloat16 *deltaK, __hip_bfloat16 *deltaV,
    __hip_bfloat16 *deltaIGatePreact, __hip_bfloat16 *deltaFGatePreact,
    __hip_bfloat16 *matC, __hip_bfloat16 *deltaH, __hip_bfloat16 *matQ,
    __hip_bfloat16 *matK, __hip_bfloat16 *matV, __hip_bfloat16 *iGatePreact,
    __hip_bfloat16 *fGatePreact, __hip_bfloat16 *vecN, __hip_bfloat16 *vecM,
    int batchSize, int numHeads, int seqLen, int dimHeads);
template void kernel_dispatchers::vlstm_bw_dispatch<__half>(
    __half *deltaQ, __half *deltaK, __half *deltaV, __half *deltaIGatePreact,
    __half *deltaFGatePreact, __half *matC, __half *deltaH, __half *matQ,
    __half *matK, __half *matV, __half *iGatePreact, __half *fGatePreact,
    __half *vecN, __half *vecM, int batchSize, int numHeads, int seqLen,
    int dimHeads);
template void kernel_dispatchers::vlstm_bw_dispatch<float>(
    float *deltaQ, float *deltaK, float *deltaV, float *deltaIGatePreact,
    float *deltaFGatePreact, float *matC, float *deltaH, float *matQ,
    float *matK, float *matV, float *iGatePreact, float *fGatePreact,
    float *vecN, float *vecM, int batchSize, int numHeads, int seqLen,
    int dimHeads);
} // namespace vlstm